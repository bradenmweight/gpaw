#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<hip/driver_types.h>
#include<hip/hip_runtime_api.h>

#include <stdio.h>
#include <time.h>

#include <sys/types.h>
#include <sys/time.h>

#include "gpaw-cuda-int.h"


#ifndef CUGPAWCOMPLEX

#define BLOCK_MAX 32
#define GRID_MAX 65535
#define BLOCK_TOTALMAX 256

static unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

#endif


extern "C" {

  void Zcuda(bmgs_translate_cuda)(Tcuda* a, const int sizea[3], 
				  const int size[3],
				  const int start1[3], const int start2[3],
#ifdef CUGPAWCOMPLEX
				  hipDoubleComplex phase,
#endif
				  enum hipMemcpyKind kind)
  {		

    if (!(size[0] && size[1] && size[2])) return;
    
    hipMemcpy3DParms myParms = {0};
    myParms.srcPtr=make_hipPitchedPtr((void*)a, sizea[2]*sizeof(Tcuda), 
				       sizea[2], sizea[1] );
    
    myParms.dstPtr=make_hipPitchedPtr((void*)a, sizea[2]*sizeof(Tcuda), 
				       sizea[2], sizea[1] );

    myParms.srcPos=make_hipPos(start1[2]*sizeof(Tcuda),start1[1],start1[0]);
    myParms.dstPos=make_hipPos(start2[2]*sizeof(Tcuda),start2[1],start2[0]);

    myParms.extent=make_hipExtent(size[2]*sizeof(Tcuda),size[1],size[0]);    
    myParms.kind=kind;
    gpaw_cudaSafeCall(hipMemcpy3D(&myParms));

#ifdef CUGPAWCOMPLEX
    Tcuda* d = a + start2[2] + (start2[1] + start2[0] * sizea[1]) * sizea[2];

    for (int i0 = 0; i0 < size[0]; i0++)
      {
	for (int i1 = 0; i1 < size[1]; i1++)
	  {
	    hipblasZscal(size[2],phase,d,1);
	    gpaw_cublasSafeCall(cublasGetError());
	    d += sizea[2];
	  }
	d += sizea[2] * (sizea[1] - size[1]);
      }
#endif
  }

}


__global__ void Zcuda(bmgs_translate_cuda_kernel)(const Tcuda* a,
						   const int3 c_sizea,
						   Tcuda* b,const int3 c_sizeb,
#ifdef CUGPAWCOMPLEX
						   hipDoubleComplex phase,
#endif		
						   int blocks,int xdiv)
{
  int xx=gridDim.x/xdiv;
  int yy=gridDim.y/blocks;
  
  int blocksi=blockIdx.y/yy;
  
  int i1=(blockIdx.y-blocksi*yy)*blockDim.y+threadIdx.y;

  int xind=blockIdx.x/xx;
  
  int i2=(blockIdx.x-xind*xx)*blockDim.x+threadIdx.x;
  
  b+=i2+(i1+(xind+blocksi*c_sizea.x)*c_sizea.y)*c_sizea.z;
  a+=i2+(i1+(xind+blocksi*c_sizea.x)*c_sizea.y)*c_sizea.z;

  while (xind<c_sizeb.x){
    if ((i2<c_sizeb.z)&&(i1<c_sizeb.y)){
#ifndef CUGPAWCOMPLEX
      b[0] = a[0];
#else
      b[0] = MULTT(phase,a[0]);
#endif
    }
    b+=xdiv*c_sizea.y*c_sizea.z;
    a+=xdiv*c_sizea.y*c_sizea.z;
    xind+=xdiv;
  }
}


extern "C" {


  void Zcuda(bmgs_translate_cuda_gpu)(Tcuda* a, const int sizea[3], 
				      const int size[3],
				      const int start1[3], const int start2[3],
#ifdef CUGPAWCOMPLEX
				      hipDoubleComplex phase, 
#endif
				      int blocks,hipStream_t stream)    
  {
    if (!(size[0] && size[1] && size[2])) return;
    
    int3 hc_sizea,hc_size;    
    hc_sizea.x=sizea[0];    hc_sizea.y=sizea[1];    hc_sizea.z=sizea[2];
    hc_size.x=size[0];    hc_size.y=size[1];    hc_size.z=size[2];

    int blockx=MIN(nextPow2(hc_size.z),BLOCK_MAX);
    int blocky=MIN(MIN(nextPow2(hc_size.y),BLOCK_TOTALMAX/blockx),BLOCK_MAX); 
    dim3 dimBlock(blockx,blocky);
    int gridx=((hc_size.z+dimBlock.x-1)/dimBlock.x);
    int xdiv=MAX(1,MIN(hc_size.x,GRID_MAX/gridx));
    int gridy=blocks*((hc_size.y+dimBlock.y-1)/dimBlock.y);    
    
    gridx=xdiv*gridx;
    dim3 dimGrid(gridx,gridy);    
    Tcuda *b=a+start2[2]+(start2[1]+start2[0]*hc_sizea.y)*hc_sizea.z;
    a+=start1[2]+(start1[1]+start1[0]*hc_sizea.y)*hc_sizea.z;

    Zcuda(bmgs_translate_cuda_kernel)<<<dimGrid, dimBlock, 0, stream>>>
      ((Tcuda*)a,hc_sizea,(Tcuda*)b,hc_size,
#ifdef CUGPAWCOMPLEX
       phase,
#endif
       blocks,xdiv);
    gpaw_cudaSafeCall(hipGetLastError());
    

  }
}


#ifndef CUGPAWCOMPLEX
#define CUGPAWCOMPLEX
#include "translate-cuda.cu"
#endif
