#include<hip/hip_runtime.h>
#include<hip/driver_types.h>
#include<hip/hip_runtime_api.h>

#include <string.h>

#include "gpaw-cuda-int.h"

extern "C" {
  void Zcuda(bmgs_cut_cuda)(const Tcuda* a, const int n[3], const int c[3],
                 Tcuda* b, const int m[3],enum hipMemcpyKind kind)
{
  /*  a += c[2] + (c[1] + c[0] * n[1]) * n[2];
  for (int i0 = 0; i0 < m[0]; i0++)
    {
      for (int i1 = 0; i1 < m[1]; i1++)
        {
          hipMemcpy(b, a, m[2] * sizeof(Tcuda),kind);
          a += n[2];
          b += m[2];
        }
      a += n[2] * (n[1] - m[1]);
    }
  */
  hipMemcpy3DParms myParms = {0};
  
  myParms.srcPtr=make_hipPitchedPtr((void*)a, n[2]*sizeof(Tcuda), n[2], n[1] );
  
  myParms.dstPtr=make_hipPitchedPtr((void*)b, m[2]*sizeof(Tcuda), m[2], m[1] );
  myParms.extent=make_hipExtent(m[2]*sizeof(Tcuda),m[1],m[0]);
  myParms.srcPos=make_hipPos(c[2]*sizeof(Tcuda),c[1],c[0]);
  
  myParms.kind=kind;
  gpaw_cudaSafeCall(hipMemcpy3D(&myParms));
}

}

#ifndef CUGPAWCOMPLEX
#define CUGPAWCOMPLEX
#include "cut-cuda.cu"
#endif
