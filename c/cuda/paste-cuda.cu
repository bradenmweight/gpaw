#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/driver_types.h>
#include<hip/hip_runtime_api.h>

#include <stdio.h>
#include <time.h>

#include <sys/types.h>
#include <sys/time.h>

#include "gpaw-cuda-int.h"

#ifdef DEBUG_CUDA
#define DEBUG_CUDA_PASTE   
#endif //DEBUG_CUDA

#ifdef DEBUG_CUDA_PASTE
extern "C" {
#include <complex.h>
  typedef double complex double_complex;
#define GPAW_MALLOC(T, n) (T*)(malloc((n) * sizeof(T)))
  void bmgs_paste(const double* a, const int n[3],
		  double* b, const int m[3], const int c[3]);
  void bmgs_pastez(const double_complex* a, const int n[3],
		   double_complex* b, const int m[3],
		   const int c[3]);
}
#endif //DEBUG_CUDA_PASTE

#ifndef CUGPAWCOMPLEX

#define BLOCK_SIZEX 32
#define BLOCK_SIZEY 16
#define BLOCK_MAX 32
#define GRID_MAX 65535
#define BLOCK_TOTALMAX 512
#define XDIV 4

static unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

#endif



extern "C" {


  void Zcuda(bmgs_paste_cuda)(const Tcuda *a, const int sizea[3],
			      Tcuda *b, const int sizeb[3], 
			      const int startb[3],int blocks,enum hipMemcpyKind kind,
			      hipStream_t stream)
  {

    if (!(sizea[0] && sizea[2] && sizea[3])) return;

    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    int ng = sizea[0] * sizea[1] * sizea[2];

    
    for (int m = 0; m < blocks; m++){            
      hipMemcpy3DParms myParms = {0};

      myParms.srcPtr=make_hipPitchedPtr((void*)(a+ng*m), sizea[2]*sizeof(Tcuda), 
					 sizea[2], sizea[1] );
      
      myParms.srcPos=make_hipPos(0*sizeof(Tcuda),0,0);
      myParms.dstPtr=make_hipPitchedPtr((void*)(b+ng2*m), sizeb[2]*sizeof(Tcuda), 
					 sizeb[2], sizeb[1] );
      
      myParms.extent=make_hipExtent(sizea[2]*sizeof(Tcuda),sizea[1],sizea[0]);
      myParms.dstPos=make_hipPos(startb[2]*sizeof(Tcuda),startb[1],startb[0]);
      
      myParms.kind=kind;


      gpaw_cudaSafeCall(hipMemcpy3DAsync(&myParms,stream));
    }
  }
}  


__global__ void Zcuda(bmgs_paste_cuda_kernel)(const double* a,
					      const int3 c_sizea,
					      double* b,const int3 c_sizeb,
					      int blocks,int xdiv)
{
  int xx=gridDim.x/xdiv;
  int yy=gridDim.y/blocks;
  
  int blocksi=blockIdx.y/yy;
  
  int i1=(blockIdx.y-blocksi*yy)*blockDim.y+threadIdx.y;

  int xind=blockIdx.x/xx;
  
  int i2=(blockIdx.x-xind*xx)*blockDim.x+threadIdx.x;
  
  b+=i2+(i1+(xind+blocksi*c_sizeb.x)*c_sizeb.y)*c_sizeb.z;
  a+=i2+(i1+(xind+blocksi*c_sizea.x)*c_sizea.y)*c_sizea.z;

  while (xind<c_sizea.x){
    if ((i2<c_sizea.z)&&(i1<c_sizea.y)){
      b[0] = a[0];
    }
    b+=xdiv*c_sizeb.y*c_sizeb.z;
    a+=xdiv*c_sizea.y*c_sizea.z;
    xind+=xdiv;
  }
}

__global__ void Zcuda(bmgs_paste_zero_cuda_kernel)(const Tcuda* a,
						   const int3 c_sizea,
						   Tcuda* b,
						   const int3 c_sizeb,
						   const int3 c_startb,
						   const int3 c_blocks_bc,
						   int blocks)
{
  int xx=gridDim.x/XDIV;
  int yy=gridDim.y/blocks;

  
  int blocksi=blockIdx.y/yy;
  int i1bl=blockIdx.y-blocksi*yy;

  int i1tid=threadIdx.y;
  int i1=i1bl*BLOCK_SIZEY+i1tid;

  int i2tid=threadIdx.x;
  
  int xind=blockIdx.x/xx;
  int i2bl=blockIdx.x-xind*xx;
  
  int i2=i2bl*BLOCK_SIZEX+i2tid;

  int xlen=(c_sizea.x+XDIV-1)/XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_sizea.x);
  
  
  b+=c_sizeb.x*c_sizeb.y*c_sizeb.z*blocksi;
  a+=c_sizea.x*c_sizea.y*c_sizea.z*blocksi;
  
  if (xind==0)  {
    Tcuda *bb=b+i2+i1*c_sizeb.z;
#pragma unroll 3
    for (int i0=0;i0<c_startb.x;i0++) {
      if ((i2<c_sizeb.z) && (i1<c_sizeb.y)) {
	bb[0]=MAKED(0);
      }
      bb+=c_sizeb.y*c_sizeb.z;
      
    }
  }
  if (xind==XDIV-1)   {
    Tcuda *bb=b+(c_startb.x+c_sizea.x)*c_sizeb.y*c_sizeb.z+i2+i1*c_sizeb.z;
#pragma unroll 3
    for (int i0=c_startb.x+c_sizea.x;i0<c_sizeb.x;i0++) {
      if ((i2<c_sizeb.z) && (i1<c_sizeb.y)) {
	bb[0]=MAKED(0);
      }
      bb+=c_sizeb.y*c_sizeb.z;
    }
  }  

  int i1blbc=gridDim.y/blocks-i1bl-1;  
  int i2blbc=gridDim.x/XDIV-i2bl-1;

  if ( i1blbc<c_blocks_bc.y || i2blbc<c_blocks_bc.z) {

    int i1bc=i1blbc*BLOCK_SIZEY+i1tid;
    int i2bc=i2blbc*BLOCK_SIZEX+i2tid;
    
    b+=(c_startb.x+xstart)*c_sizeb.y*c_sizeb.z;
    for (int i0=xstart;i0<xend;i0++) {	      
      if ((i1bc<c_startb.y) && (i2<c_sizeb.z)){
	b[i2+i1bc*c_sizeb.z]=MAKED(0);
      }
      if ((i1bc+c_sizea.y+c_startb.y<c_sizeb.y) && (i2<c_sizeb.z)){
	b[i2+i1bc*c_sizeb.z+(c_sizea.y+c_startb.y)*c_sizeb.z]=MAKED(0);
      }
      if ((i2bc<c_startb.z) && (i1<c_sizeb.y)){
	b[i2bc+i1*c_sizeb.z]=MAKED(0);
      }
      if ((i2bc+c_sizea.z+c_startb.z<c_sizeb.z) && (i1<c_sizeb.y)){
	b[i2bc+i1*c_sizeb.z+c_sizea.z+c_startb.z]=MAKED(0);
      }
      b+=c_sizeb.y*c_sizeb.z;
    }    
  }else{
    
    b+=c_startb.z+(c_startb.y+c_startb.x*c_sizeb.y)*c_sizeb.z;
    
    b+=i2+i1*c_sizeb.z+xstart*c_sizeb.y*c_sizeb.z;
    a+=i2+i1*c_sizea.z+xstart*c_sizea.y*c_sizea.z;
    for (int i0=xstart;i0<xend;i0++) {	
      if ((i2<c_sizea.z)&&(i1<c_sizea.y)){
	b[0] = a[0];
      }
      b+=c_sizeb.y*c_sizeb.z;
      a+=c_sizea.y*c_sizea.z;        
    }
  }
}


extern "C" {

  
  void Zcuda(bmgs_paste_cuda_gpu)(const Tcuda* a, const int sizea[3],
				  Tcuda* b, const int sizeb[3], 
				  const int startb[3],
				  int blocks,hipStream_t stream)
  {
    if (!(sizea[0] && sizea[1] && sizea[2])) return;    


      
    int3 hc_sizea,hc_sizeb;
    hc_sizea.x=sizea[0];    hc_sizea.y=sizea[1];    hc_sizea.z=sizea[2]*sizeof(Tcuda)/sizeof(double);
    hc_sizeb.x=sizeb[0];    hc_sizeb.y=sizeb[1];    hc_sizeb.z=sizeb[2]*sizeof(Tcuda)/sizeof(double);

    
#ifdef DEBUG_CUDA_PASTE
#ifndef CUGPAWCOMPLEX      
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    int ng = sizea[0] * sizea[1] * sizea[2];
#else
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2] * 2;
    int ng = sizea[0] * sizea[1] * sizea[2] * 2;
#endif //CUGPAWCOMPLEX      
    double* b_cpu=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu=GPAW_MALLOC(double, ng*blocks);
    double* b_cpu2=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu2=GPAW_MALLOC(double, ng*blocks);
    Tcuda* b2=b;

    GPAW_CUDAMEMCPY(a_cpu,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu,b,double, ng2*blocks, hipMemcpyDeviceToHost);
#endif //DEBUG_CUDA_PASTE

    int blockx=MIN(nextPow2(hc_sizea.z),BLOCK_MAX);
    int blocky=MIN(MIN(nextPow2(hc_sizea.y),BLOCK_TOTALMAX/blockx),BLOCK_MAX); 
    dim3 dimBlock(blockx,blocky);
    int gridx=((hc_sizea.z+dimBlock.x-1)/dimBlock.x);
    int xdiv=MAX(1,MIN(hc_sizea.x,GRID_MAX/gridx));
    int gridy=blocks*((hc_sizea.y+dimBlock.y-1)/dimBlock.y);    

    gridx=xdiv*gridx;
    dim3 dimGrid(gridx,gridy);    
    b+=startb[2]+(startb[1]+startb[0]*sizeb[1])*sizeb[2];      
    Zcuda(bmgs_paste_cuda_kernel)<<<dimGrid, dimBlock, 0, stream>>>
      ((double*)a,hc_sizea,(double*)b,hc_sizeb,blocks,xdiv);
    gpaw_cudaSafeCall(hipGetLastError());

#ifdef DEBUG_CUDA_PASTE
    for (int m = 0; m < blocks; m++){            
#ifndef CUGPAWCOMPLEX      
      bmgs_paste(a_cpu + m * ng, sizea, b_cpu + m * ng2,
		 sizeb, startb);
#else
      bmgs_pastez((const double_complex*)(a_cpu + m * ng), sizea,
		  (double_complex*)(b_cpu + m * ng2),
		  sizeb, startb);
#endif //CUGPAWCOMPLEX
    }
    hipDeviceSynchronize();
    GPAW_CUDAMEMCPY(a_cpu2,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu2,b2,double, ng2*blocks, hipMemcpyDeviceToHost);
    double a_err=0;
    double b_err=0;
    for (int i=0;i<ng2*blocks;i++) {      
      b_err=MAX(b_err,fabs(b_cpu[i]-b_cpu2[i]));
      if (i<ng*blocks){
	a_err=MAX(a_err,fabs(a_cpu[i]-a_cpu2[i]));
      }
    }
    if ((b_err>GPAW_CUDA_ABS_TOL_EXCT) || (a_err>GPAW_CUDA_ABS_TOL_EXCT)){
      fprintf(stderr,"Debug cuda paste errors: a %g b %g\n",a_err,b_err); fflush(stderr);
    }
    free(a_cpu);
    free(b_cpu);
    free(a_cpu2);
    free(b_cpu2);
#endif //DEBUG_CUDA_PASTE
    
  }
  

  void Zcuda(bmgs_paste_zero_cuda_gpu)(const Tcuda* a, const int sizea[3],
				       Tcuda* b, const int sizeb[3], 
				       const int startb[3],
				       int blocks,hipStream_t stream)
  {
    if (!(sizea[0] && sizea[1] && sizea[2])) return;
    
    int3 hc_sizea,hc_sizeb,hc_startb;    
    hc_sizea.x=sizea[0];    hc_sizea.y=sizea[1];    hc_sizea.z=sizea[2];
    hc_sizeb.x=sizeb[0];    hc_sizeb.y=sizeb[1];    hc_sizeb.z=sizeb[2];
    hc_startb.x=startb[0];    hc_startb.y=startb[1];    hc_startb.z=startb[2];

    int3 bc_blocks;

#ifdef DEBUG_CUDA_PASTE
#ifndef CUGPAWCOMPLEX      
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    int ng = sizea[0] * sizea[1] * sizea[2];
#else
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2] * 2;
    int ng = sizea[0] * sizea[1] * sizea[2] * 2;
#endif //CUGPAWCOMPLEX      
    double* b_cpu=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu=GPAW_MALLOC(double, ng*blocks);
    double* b_cpu2=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu2=GPAW_MALLOC(double, ng*blocks);

    GPAW_CUDAMEMCPY(a_cpu,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu,b,double, ng2*blocks, hipMemcpyDeviceToHost);
#endif //DEBUG_CUDA_PASTE


    bc_blocks.y=hc_sizeb.y-hc_sizea.y>0 ? 
      MAX((hc_sizeb.y-hc_sizea.y+BLOCK_SIZEY-1)/BLOCK_SIZEY,1) : 0;
    bc_blocks.z=hc_sizeb.z-hc_sizea.z>0 ?
      MAX((hc_sizeb.z-hc_sizea.z+BLOCK_SIZEX-1)/BLOCK_SIZEX,1) : 0;
    
    int gridy=blocks*((sizeb[1]+BLOCK_SIZEY-1)/BLOCK_SIZEY+bc_blocks.y);
    
    int gridx=XDIV*((sizeb[2]+BLOCK_SIZEX-1)/BLOCK_SIZEX+bc_blocks.z);
    

    dim3 dimBlock(BLOCK_SIZEX,BLOCK_SIZEY); 
    dim3 dimGrid(gridx,gridy);    
    
    //    b+=startb[2]+(startb[1]+startb[0]*hc_sizeb.y)*hc_sizeb.z;
    Zcuda(bmgs_paste_zero_cuda_kernel)<<<dimGrid, dimBlock, 0, stream>>>
      ((Tcuda*)a,hc_sizea,(Tcuda*)b,hc_sizeb,hc_startb,bc_blocks,blocks);
    
    gpaw_cudaSafeCall(hipGetLastError());
    
#ifdef DEBUG_CUDA_PASTE    
    for (int m = 0; m < blocks; m++){            
      memset(b_cpu + m * ng2, 0, ng2 * sizeof(double));
#ifndef CUGPAWCOMPLEX      
      bmgs_paste(a_cpu + m * ng, sizea, b_cpu + m * ng2,
		 sizeb, startb);
#else
      bmgs_pastez((const double_complex*)(a_cpu + m * ng), sizea,
		  (double_complex*)(b_cpu + m * ng2),
		  sizeb, startb);
#endif //CUGPAWCOMPLEX
    }
    hipDeviceSynchronize();
    GPAW_CUDAMEMCPY(a_cpu2,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu2,b,double, ng2*blocks, hipMemcpyDeviceToHost);
    double a_err=0;
    double b_err=0;
    for (int i=0;i<ng2*blocks;i++) {      
      b_err=MAX(b_err,fabs(b_cpu[i]-b_cpu2[i]));
      if (i<ng*blocks){
	a_err=MAX(a_err,fabs(a_cpu[i]-a_cpu2[i]));
      }
    }
    if ((b_err>GPAW_CUDA_ABS_TOL_EXCT) || (a_err>GPAW_CUDA_ABS_TOL_EXCT)){
      fprintf(stderr,"Debug cuda paste_zero errors: a %g b %g\n",a_err,b_err);
    }
    free(a_cpu);
    free(b_cpu);
    free(a_cpu2);
    free(b_cpu2);
#endif //DEBUG_CUDA_PASTE
    
  }
}

#ifndef CUGPAWCOMPLEX
#define CUGPAWCOMPLEX
#include "paste-cuda.cu"

extern "C" {
  double bmgs_paste_cuda_cpu(const double* a, const int sizea[3],
			     double* b, const int sizeb[3], 
			     const int startb[3])
  {
    double *adev,*bdev;
    
    struct timeval  t0, t1; 
    double flops;
    int asize=sizea[0]*sizea[1]*sizea[2];
    int bsize=sizeb[0]*sizeb[1]*sizeb[2];
    
    
    
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));
    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    
    gettimeofday(&t0,NULL);  
    bmgs_paste_cuda_gpu(adev, sizea,
			bdev, sizeb, startb,1,0);
    
    
    hipDeviceSynchronize();
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
       
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));
    
    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 
    
    return flops;
    }


  double bmgs_paste_zero_cuda_cpu(const double* a, const int sizea[3],
				   double* b, const int sizeb[3], 
				   const int startb[3])
  {
    double *adev,*bdev;
    
    struct timeval  t0, t1; 
    double flops;
    int asize=sizea[0]*sizea[1]*sizea[2];
    int bsize=sizeb[0]*sizeb[1]*sizeb[2];
    
    
    
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));
    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    
    gettimeofday(&t0,NULL);  
    bmgs_paste_zero_cuda_gpu(adev, sizea,
			     bdev, sizeb, startb,1,0);
    
    
    hipDeviceSynchronize();
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
    
        
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));
    
    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 
    
    return flops;
    }
}

#endif
