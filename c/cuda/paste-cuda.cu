#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/driver_types.h>
#include<hip/hip_runtime_api.h>

#include <stdio.h>
#include <time.h>

#include <sys/types.h>
#include <sys/time.h>

#include "gpaw-cuda-int.h"

#ifdef DEBUG_CUDA
#define DEBUG_CUDA_PASTE   
#endif //DEBUG_CUDA

#ifdef DEBUG_CUDA_PASTE
extern "C" {
#include <complex.h>
  typedef double complex double_complex;
#define GPAW_MALLOC(T, n) (T*)(malloc((n) * sizeof(T)))
  void bmgs_paste(const double* a, const int n[3],
		  double* b, const int m[3], const int c[3]);
  void bmgs_pastez(const double_complex* a, const int n[3],
		   double_complex* b, const int m[3],
		   const int c[3]);
}
#endif //DEBUG_CUDA_PASTE

#ifndef CUGPAWCOMPLEX

#define BLOCK_SIZEX 32
#define BLOCK_SIZEY 8
#define XDIV 4

#endif


extern "C" {


  void Zcuda(bmgs_paste_cuda)(const Tcuda *a, const int sizea[3],
			      Tcuda *b, const int sizeb[3], 
			      const int startb[3],enum hipMemcpyKind kind)
  {

    if (!(sizea[0] && sizea[2] && sizea[3])) return;

    hipMemcpy3DParms myParms = {0};
    
    myParms.srcPtr=make_hipPitchedPtr((void*)a, sizea[2]*sizeof(Tcuda), 
				       sizea[2], sizea[1] );
    
    myParms.dstPtr=make_hipPitchedPtr((void*)b, sizeb[2]*sizeof(Tcuda), 
				       sizeb[2], sizeb[1] );
    myParms.extent=make_hipExtent(sizea[2]*sizeof(Tcuda),sizea[1],sizea[0]);
    myParms.dstPos=make_hipPos(startb[2]*sizeof(Tcuda),startb[1],startb[0]);
    
    myParms.kind=kind;
    gpaw_cudaSafeCall(hipMemcpy3D(&myParms));
  }
}  


__global__ void Zcuda(bmgs_paste_cuda_kernel)(const Tcuda* a,
					      const int3 c_sizea,
					      Tcuda* b,const int3 c_sizeb,
					      int blocks)
{
  int xx=gridDim.x/XDIV;
  int yy=gridDim.y/blocks;
  
  int blocksi=blockIdx.y/yy;
  int i1bl=blockIdx.y-blocksi*yy;
  
  int i1tid=threadIdx.y;
  int i1=i1bl*BLOCK_SIZEY+i1tid;

  int xind=blockIdx.x/xx;
  int i2bl=blockIdx.x-xind*xx;
  
  int i2=i2bl*BLOCK_SIZEX+threadIdx.x;
  
  int xlen=(c_sizea.x+XDIV-1)/XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_sizea.x);
  
  b+=c_sizeb.x*c_sizeb.y*c_sizeb.z*blocksi;
  a+=c_sizea.x*c_sizea.y*c_sizea.z*blocksi;


  b+=i2+i1*c_sizeb.z+xstart*c_sizeb.y*c_sizeb.z;
  a+=i2+i1*c_sizea.z+xstart*c_sizea.y*c_sizea.z;
  for (int i0=xstart;i0<xend;i0++) {	
    if ((i2<c_sizea.z)&&(i1<c_sizea.y)){
      b[0] = a[0];
    }
    b+=c_sizeb.y*c_sizeb.z;
    a+=c_sizea.y*c_sizea.z;        
  }
}

__global__ void Zcuda(bmgs_paste_zero_cuda_kernel)(const Tcuda* a,
						   const int3 c_sizea,
						   Tcuda* b,
						   const int3 c_sizeb,
						   const int3 c_startb,
						   const int3 c_blocks_bc,
						   int blocks)
{
  int xx=gridDim.x/XDIV;
  int yy=gridDim.y/blocks;

  
  int blocksi=blockIdx.y/yy;
  int i1bl=blockIdx.y-blocksi*yy;

  int i1tid=threadIdx.y;
  int i1=i1bl*BLOCK_SIZEY+i1tid;

  int i2tid=threadIdx.x;
  
  int xind=blockIdx.x/xx;
  int i2bl=blockIdx.x-xind*xx;
  
  int i2=i2bl*BLOCK_SIZEX+i2tid;

  int xlen=(c_sizea.x+XDIV-1)/XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_sizea.x);
  
  
  b+=c_sizeb.x*c_sizeb.y*c_sizeb.z*blocksi;
  a+=c_sizea.x*c_sizea.y*c_sizea.z*blocksi;
  
  if (xind==0)  {
    Tcuda *bb=b+i2+i1*c_sizeb.z;
#pragma unroll 3
    for (int i0=0;i0<c_startb.x;i0++) {
      if ((i2<c_sizeb.z) && (i1<c_sizeb.y)) {
	bb[0]=MAKED(0);
      }
      bb+=c_sizeb.y*c_sizeb.z;
      
    }
  }
  if (xind==XDIV-1)   {
    Tcuda *bb=b+(c_startb.x+c_sizea.x)*c_sizeb.y*c_sizeb.z+i2+i1*c_sizeb.z;
#pragma unroll 3
    for (int i0=c_startb.x+c_sizea.x;i0<c_sizeb.x;i0++) {
      if ((i2<c_sizeb.z) && (i1<c_sizeb.y)) {
	bb[0]=MAKED(0);
      }
      bb+=c_sizeb.y*c_sizeb.z;
    }
  }  

  int i1blbc=gridDim.y/blocks-i1bl-1;  
  int i2blbc=gridDim.x/XDIV-i2bl-1;

  if ( i1blbc<c_blocks_bc.y || i2blbc<c_blocks_bc.z) {

    int i1bc=i1blbc*BLOCK_SIZEY+i1tid;
    int i2bc=i2blbc*BLOCK_SIZEX+i2tid;
    
    b+=(c_startb.x+xstart)*c_sizeb.y*c_sizeb.z;
    for (int i0=xstart;i0<xend;i0++) {	      
      if ((i1bc<c_startb.y) && (i2<c_sizeb.z)){
	b[i2+i1bc*c_sizeb.z]=MAKED(0);
      }
      if ((i1bc+c_sizea.y+c_startb.y<c_sizeb.y) && (i2<c_sizeb.z)){
	b[i2+i1bc*c_sizeb.z+(c_sizea.y+c_startb.y)*c_sizeb.z]=MAKED(0);
      }
      if ((i2bc<c_startb.z) && (i1<c_sizeb.y)){
	b[i2bc+i1*c_sizeb.z]=MAKED(0);
      }
      if ((i2bc+c_sizea.z+c_startb.z<c_sizeb.z) && (i1<c_sizeb.y)){
	b[i2bc+i1*c_sizeb.z+c_sizea.z+c_startb.z]=MAKED(0);
      }
      b+=c_sizeb.y*c_sizeb.z;
    }    
  }else{
    
    b+=c_startb.z+(c_startb.y+c_startb.x*c_sizeb.y)*c_sizeb.z;
    
    b+=i2+i1*c_sizeb.z+xstart*c_sizeb.y*c_sizeb.z;
    a+=i2+i1*c_sizea.z+xstart*c_sizea.y*c_sizea.z;
    for (int i0=xstart;i0<xend;i0++) {	
      if ((i2<c_sizea.z)&&(i1<c_sizea.y)){
	b[0] = a[0];
      }
      b+=c_sizeb.y*c_sizeb.z;
      a+=c_sizea.y*c_sizea.z;        
    }
  }
}


extern "C" {
  
  void Zcuda(bmgs_paste_cuda_gpu)(const Tcuda* a, const int sizea[3],
				  Tcuda* b, const int sizeb[3], 
				  const int startb[3],
				  int blocks,hipStream_t stream)
  {
    if (!(sizea[0] && sizea[1] && sizea[2])) return;    

    int3 hc_sizea,hc_sizeb;    
    hc_sizea.x=sizea[0];    hc_sizea.y=sizea[1];    hc_sizea.z=sizea[2];
    hc_sizeb.x=sizeb[0];    hc_sizeb.y=sizeb[1];    hc_sizeb.z=sizeb[2];
    
#ifdef DEBUG_CUDA_PASTE
#ifndef CUGPAWCOMPLEX      
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    int ng = sizea[0] * sizea[1] * sizea[2];
#else
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2] * 2;
    int ng = sizea[0] * sizea[1] * sizea[2] * 2;
#endif //CUGPAWCOMPLEX      
    double* b_cpu=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu=GPAW_MALLOC(double, ng*blocks);
    double* b_cpu2=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu2=GPAW_MALLOC(double, ng*blocks);
    Tcuda* b2=b;

    GPAW_CUDAMEMCPY(a_cpu,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu,b,double, ng2*blocks, hipMemcpyDeviceToHost);
#endif //DEBUG_CUDA_PASTE


    int gridy=blocks*((sizea[1]+BLOCK_SIZEY-1)/BLOCK_SIZEY);    
    int gridx=XDIV*((sizea[2]+BLOCK_SIZEX-1)/BLOCK_SIZEX);
    
    
    dim3 dimBlock(BLOCK_SIZEX,BLOCK_SIZEY); 
    dim3 dimGrid(gridx,gridy);    

    b+=startb[2]+(startb[1]+startb[0]*hc_sizeb.y)*hc_sizeb.z;
    Zcuda(bmgs_paste_cuda_kernel)<<<dimGrid, dimBlock, 0, stream>>>
      ((Tcuda*)a,hc_sizea,(Tcuda*)b,hc_sizeb,blocks);
    
    gpaw_cudaSafeCall(hipGetLastError());

#ifdef DEBUG_CUDA_PASTE
    for (int m = 0; m < blocks; m++){            
#ifndef CUGPAWCOMPLEX      
      bmgs_paste(a_cpu + m * ng, sizea, b_cpu + m * ng2,
		 sizeb, startb);
#else
      bmgs_pastez((const double_complex*)(a_cpu + m * ng), sizea,
		  (double_complex*)(b_cpu + m * ng2),
		  sizeb, startb);
#endif //CUGPAWCOMPLEX
    }
    hipDeviceSynchronize();
    GPAW_CUDAMEMCPY(a_cpu2,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu2,b2,double, ng2*blocks, hipMemcpyDeviceToHost);
    double a_err=0;
    double b_err=0;
    for (int i=0;i<ng2*blocks;i++) {      
      b_err=MAX(b_err,fabs(b_cpu[i]-b_cpu2[i]));
      if (i<ng*blocks){
	a_err=MAX(a_err,fabs(a_cpu[i]-a_cpu2[i]));
      }
    }
    if ((b_err>GPAW_CUDA_ABS_TOL_EXCT) || (a_err>GPAW_CUDA_ABS_TOL_EXCT)){
      fprintf(stderr,"Debug cuda paste errors: a %g b %g\n",a_err,b_err);
    }
    free(a_cpu);
    free(b_cpu);
    free(a_cpu2);
    free(b_cpu2);
#endif //DEBUG_CUDA_PASTE
    
  }
  

  void Zcuda(bmgs_paste_zero_cuda_gpu)(const Tcuda* a, const int sizea[3],
				       Tcuda* b, const int sizeb[3], 
				       const int startb[3],
				       int blocks,hipStream_t stream)
  {
    if (!(sizea[0] && sizea[1] && sizea[2])) return;
    
    int3 hc_sizea,hc_sizeb,hc_startb;    
    hc_sizea.x=sizea[0];    hc_sizea.y=sizea[1];    hc_sizea.z=sizea[2];
    hc_sizeb.x=sizeb[0];    hc_sizeb.y=sizeb[1];    hc_sizeb.z=sizeb[2];
    hc_startb.x=startb[0];    hc_startb.y=startb[1];    hc_startb.z=startb[2];

    int3 bc_blocks;

#ifdef DEBUG_CUDA_PASTE
#ifndef CUGPAWCOMPLEX      
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2];
    int ng = sizea[0] * sizea[1] * sizea[2];
#else
    int ng2 = sizeb[0] * sizeb[1] * sizeb[2] * 2;
    int ng = sizea[0] * sizea[1] * sizea[2] * 2;
#endif //CUGPAWCOMPLEX      
    double* b_cpu=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu=GPAW_MALLOC(double, ng*blocks);
    double* b_cpu2=GPAW_MALLOC(double, ng2*blocks);
    double* a_cpu2=GPAW_MALLOC(double, ng*blocks);

    GPAW_CUDAMEMCPY(a_cpu,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu,b,double, ng2*blocks, hipMemcpyDeviceToHost);
#endif //DEBUG_CUDA_PASTE


    bc_blocks.y=hc_sizeb.y-hc_sizea.y>0 ? 
      MAX((hc_sizeb.y-hc_sizea.y+BLOCK_SIZEY-1)/BLOCK_SIZEY,1) : 0;
    bc_blocks.z=hc_sizeb.z-hc_sizea.z>0 ?
      MAX((hc_sizeb.z-hc_sizea.z+BLOCK_SIZEX-1)/BLOCK_SIZEX,1) : 0;
    
    int gridy=blocks*((sizeb[1]+BLOCK_SIZEY-1)/BLOCK_SIZEY+bc_blocks.y);
    
    int gridx=XDIV*((sizeb[2]+BLOCK_SIZEX-1)/BLOCK_SIZEX+bc_blocks.z);
    

    dim3 dimBlock(BLOCK_SIZEX,BLOCK_SIZEY); 
    dim3 dimGrid(gridx,gridy);    
    
    //    b+=startb[2]+(startb[1]+startb[0]*hc_sizeb.y)*hc_sizeb.z;
    Zcuda(bmgs_paste_zero_cuda_kernel)<<<dimGrid, dimBlock, 0, stream>>>
      ((Tcuda*)a,hc_sizea,(Tcuda*)b,hc_sizeb,hc_startb,bc_blocks,blocks);
    
    gpaw_cudaSafeCall(hipGetLastError());
    
#ifdef DEBUG_CUDA_PASTE    
    for (int m = 0; m < blocks; m++){            
      memset(b_cpu + m * ng2, 0, ng2 * sizeof(double));
#ifndef CUGPAWCOMPLEX      
      bmgs_paste(a_cpu + m * ng, sizea, b_cpu + m * ng2,
		 sizeb, startb);
#else
      bmgs_pastez((const double_complex*)(a_cpu + m * ng), sizea,
		  (double_complex*)(b_cpu + m * ng2),
		  sizeb, startb);
#endif //CUGPAWCOMPLEX
    }
    hipDeviceSynchronize();
    GPAW_CUDAMEMCPY(a_cpu2,a,double, ng*blocks, hipMemcpyDeviceToHost);
    GPAW_CUDAMEMCPY(b_cpu2,b,double, ng2*blocks, hipMemcpyDeviceToHost);
    double a_err=0;
    double b_err=0;
    for (int i=0;i<ng2*blocks;i++) {      
      b_err=MAX(b_err,fabs(b_cpu[i]-b_cpu2[i]));
      if (i<ng*blocks){
	a_err=MAX(a_err,fabs(a_cpu[i]-a_cpu2[i]));
      }
    }
    if ((b_err>GPAW_CUDA_ABS_TOL_EXCT) || (a_err>GPAW_CUDA_ABS_TOL_EXCT)){
      fprintf(stderr,"Debug cuda paste_zero errors: a %g b %g\n",a_err,b_err);
    }
    free(a_cpu);
    free(b_cpu);
    free(a_cpu2);
    free(b_cpu2);
#endif //DEBUG_CUDA_PASTE
    
  }
}

#ifndef CUGPAWCOMPLEX
#define CUGPAWCOMPLEX
#include "paste-cuda.cu"

extern "C" {
  double bmgs_paste_cuda_cpu(const double* a, const int sizea[3],
			     double* b, const int sizeb[3], 
			     const int startb[3])
  {
    double *adev,*bdev;
    
    struct timeval  t0, t1; 
    double flops;
    int asize=sizea[0]*sizea[1]*sizea[2];
    int bsize=sizeb[0]*sizeb[1]*sizeb[2];
    
    
    
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));
    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    
    gettimeofday(&t0,NULL);  
    bmgs_paste_cuda_gpu(adev, sizea,
			bdev, sizeb, startb,1,0);
    
    
    hipDeviceSynchronize();
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
       
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));
    
    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 
    
    return flops;
    }


  double bmgs_paste_zero_cuda_cpu(const double* a, const int sizea[3],
				   double* b, const int sizeb[3], 
				   const int startb[3])
  {
    double *adev,*bdev;
    
    struct timeval  t0, t1; 
    double flops;
    int asize=sizea[0]*sizea[1]*sizea[2];
    int bsize=sizeb[0]*sizeb[1]*sizeb[2];
    
    
    
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));
    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    
    gettimeofday(&t0,NULL);  
    bmgs_paste_zero_cuda_gpu(adev, sizea,
			     bdev, sizeb, startb,1,0);
    
    
    hipDeviceSynchronize();
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
    
        
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));
    
    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 
    
    return flops;
    }
}

#endif
