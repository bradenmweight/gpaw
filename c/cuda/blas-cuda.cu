#include <Python.h>
#define PY_ARRAY_UNIQUE_SYMBOL GPAW_ARRAY_API
#define NO_IMPORT_ARRAY
#include <numpy/arrayobject.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>

#include <gpaw-cuda-int.h>

extern "C" {


PyObject* scal_cuda_gpu(PyObject *self, PyObject *args)
{
  Py_complex alpha;

  hipDeviceptr_t x_gpu;
  PyObject *x_shape;
  PyArray_Descr *type; 


  if (!PyArg_ParseTuple(args, "DnOO", &alpha, &x_gpu,&x_shape,&type))
    return NULL;

  int n = PyInt_AsLong(PyTuple_GetItem(x_shape,0));
  Py_ssize_t nd=PyTuple_Size(x_shape);
  for (int d = 1; d < nd; d++)
    n *= PyInt_AsLong(PyTuple_GetItem(x_shape,d));
  int incx = 1;
  if (type->type_num == PyArray_DOUBLE)
    hipblasDscal(n, alpha.real,
		(double*)x_gpu, incx);		
  else {
    hipDoubleComplex alpha_gpu={alpha.real,alpha.imag};
    hipblasZscal(n, alpha_gpu,
		(hipDoubleComplex*)x_gpu, incx);
  }
  gpaw_cublasSafeCall(cublasGetError());

  Py_RETURN_NONE;
}




PyObject* gemm_cuda_gpu(PyObject *self, PyObject *args)
{
  Py_complex alpha;
  Py_complex beta;

  hipDeviceptr_t a_gpu;
  hipDeviceptr_t b_gpu;
  hipDeviceptr_t c_gpu;
  PyObject *a_shape,*b_shape,*c_shape;
  PyArray_Descr *type; 

  char transa = 'n';
  if (!PyArg_ParseTuple(args, "DnOnODnOO|c", &alpha, &a_gpu,&a_shape, &b_gpu,
			&b_shape, &beta, &c_gpu,&c_shape,&type,&transa))
    return NULL;
  int m, k, lda, ldb, ldc;

  int n = PyInt_AsLong(PyTuple_GetItem(b_shape,0));

  if (transa == 'n')
    {
      m = PyInt_AsLong(PyTuple_GetItem(a_shape,1));
      
      for (int i = 2; i < PyTuple_Size(a_shape); i++)
	m *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));
      k = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
      lda = m;
      ldb = k;
      ldc = m;
    }
  else
    {
      k = PyInt_AsLong(PyTuple_GetItem(a_shape,1));
      for (int i = 2; i < PyTuple_Size(a_shape); i++)
	k *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));
      m = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
      
      lda = k;
      ldb = k;
      ldc = m;
      
    }
  
  if (type->type_num == PyArray_DOUBLE)
    hipblasDgemm(transa, 'n', m, n, k, 
		alpha.real,(double*)a_gpu ,lda, (double*)b_gpu, ldb, 
		beta.real, (double*)c_gpu, ldc);

  else {
    hipDoubleComplex alpha_gpu={alpha.real,alpha.imag};
    hipDoubleComplex beta_gpu={beta.real,beta.imag};
    hipblasZgemm(transa, 'n', m, n, k, 
		alpha_gpu,
		(hipDoubleComplex*)a_gpu ,lda,
		(hipDoubleComplex*)b_gpu, ldb, 
		beta_gpu,
		(hipDoubleComplex*)c_gpu, ldc);
  }
  
  gpaw_cublasSafeCall(cublasGetError());
  Py_RETURN_NONE;
}


PyObject* gemv_cuda_gpu(PyObject *self, PyObject *args)
{
  Py_complex alpha;

  hipDeviceptr_t a_gpu;
  hipDeviceptr_t x_gpu;
  hipDeviceptr_t y_gpu;

  Py_complex beta;
  PyObject *a_shape,*x_shape;
  PyArray_Descr *type;

  char trans = 't';
  if (!PyArg_ParseTuple(args, "DnOnODn0|c", &alpha, &a_gpu,&a_shape, &x_gpu,&x_shape, &beta, &y_gpu,&type,&trans))
    return NULL;

  int m, n, lda, incx, incy;

  if (trans == 'n')
    {
      m = PyInt_AsLong(PyTuple_GetItem(a_shape,1));
      
      for (int i = 2; i < PyTuple_Size(a_shape); i++)
	m *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));
      n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
      lda = m;
    }
  else
    {
      n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
      for (int i = 1; i < PyTuple_Size(a_shape)-1; i++)
	n *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));
      m = PyInt_AsLong(PyTuple_GetItem(a_shape,PyTuple_Size(a_shape)-1));
      
      lda = m;

    }


  incx = 1;
  incy = 1;

  if (type->type_num == PyArray_DOUBLE)
    hipblasDgemv(trans, m, n, 
		alpha.real,(double*)a_gpu ,lda, (double*)x_gpu, incx, 
		beta.real, (double*)y_gpu, incy);
  else{
    hipDoubleComplex alpha_gpu={alpha.real,alpha.imag};
    hipDoubleComplex beta_gpu={beta.real,beta.imag};
    hipblasZgemv(trans, m, n, 
		alpha_gpu,
		(hipDoubleComplex*)a_gpu ,lda,
		(hipDoubleComplex*)x_gpu, incx, 
		beta_gpu,
		(hipDoubleComplex*)y_gpu, incy);
  }

  Py_RETURN_NONE;
}




PyObject* axpy_cuda_gpu(PyObject *self, PyObject *args)
{
  Py_complex alpha;

  hipDeviceptr_t x_gpu;
  hipDeviceptr_t y_gpu;
  PyObject *x_shape,*y_shape;
  PyArray_Descr *type; 


  if (!PyArg_ParseTuple(args, "DnOnOO", &alpha, &x_gpu,&x_shape, &y_gpu,
			&y_shape,&type))
    return NULL;

  int n = PyInt_AsLong(PyTuple_GetItem(x_shape,0));
  Py_ssize_t nd=PyTuple_Size(x_shape);
  for (int d = 1; d < nd; d++)
    n *= PyInt_AsLong(PyTuple_GetItem(x_shape,d));
  int incx = 1;
  int incy = 1;
  if (type->type_num == PyArray_DOUBLE)
    hipblasDaxpy(n, alpha.real,
		(double*)x_gpu, incx,
		(double*)y_gpu, incy);
  else {
    hipDoubleComplex alpha_gpu={alpha.real,alpha.imag};
    hipblasZaxpy(n, alpha_gpu,
		(hipDoubleComplex*)x_gpu, incx,
		(hipDoubleComplex*)y_gpu, incy);
  }
  gpaw_cublasSafeCall(cublasGetError());

  
  Py_RETURN_NONE;
}


PyObject* rk_cuda_gpu(PyObject *self, PyObject *args)
{
  double alpha;

  double beta;
  
  hipDeviceptr_t a_gpu;
  hipDeviceptr_t c_gpu;
  PyObject *a_shape,*c_shape;
  PyArray_Descr *type; 


  if (!PyArg_ParseTuple(args, "dnOdnOO", &alpha, &a_gpu,&a_shape, &beta, 
			&c_gpu,&c_shape,&type))
    return NULL;


  int n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
  int k = PyInt_AsLong(PyTuple_GetItem(a_shape,1));

  for (int d = 2; d < PyTuple_Size(a_shape); d++)
    k *= PyInt_AsLong(PyTuple_GetItem(a_shape,d));
  int ldc = n;
  if (type->type_num == PyArray_DOUBLE)
    hipblasDsyrk('u', 't', n, k,
		alpha, (double*)a_gpu, k, beta,
		(double*)c_gpu, ldc);
  else {
    hipblasZherk('u', 't', n, k,
		alpha, (hipDoubleComplex*)a_gpu, k,
		beta, (hipDoubleComplex*)c_gpu, ldc);
  }
  gpaw_cublasSafeCall(cublasGetError());

  Py_RETURN_NONE;
}


PyObject* r2k_cuda_gpu(PyObject *self, PyObject *args)
{
  Py_complex alpha;
  double beta;
  
  hipDeviceptr_t a_gpu;
  hipDeviceptr_t b_gpu;
  hipDeviceptr_t c_gpu;
  PyObject *a_shape,*b_shape,*c_shape;
  PyArray_Descr *type; 


  if (!PyArg_ParseTuple(args, "DnOnOdnOO", &alpha, &a_gpu,&a_shape,&b_gpu,
			&b_shape, &beta, &c_gpu,&c_shape,&type))
    return NULL;

  int n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
  int k = PyInt_AsLong(PyTuple_GetItem(a_shape,1));

  for (int d = 2; d < PyTuple_Size(a_shape); d++)
    k *= PyInt_AsLong(PyTuple_GetItem(a_shape,d));

  int ldc = n;
  
  if (type->type_num == PyArray_DOUBLE)
    hipblasDsyr2k('u', 't', n, k,
		 alpha.real, (double*)a_gpu, k, (double*)b_gpu,k,beta,
		 (double*)c_gpu, ldc);
  else {
    hipDoubleComplex alpha_gpu={alpha.real,alpha.imag};
    hipblasZher2k('u', 't', n, k,
		 alpha_gpu, 
		 (hipDoubleComplex*)a_gpu, k, 
		 (hipDoubleComplex*)b_gpu,k,
		 beta, 
		 (hipDoubleComplex*)c_gpu, ldc);
  }
  gpaw_cublasSafeCall(cublasGetError());


  Py_RETURN_NONE;
}


PyObject* dotc_cuda_gpu(PyObject *self, PyObject *args)
{
  hipDeviceptr_t a_gpu;
  hipDeviceptr_t b_gpu;
  
  PyObject *a_shape;
  PyArray_Descr *type;


  if (!PyArg_ParseTuple(args, "nOnO", &a_gpu,&a_shape,&b_gpu,&type))
    return NULL;

  int n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
  
  for (int i = 1; i < PyTuple_Size(a_shape); i++)
    n *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));

  int incx = 1;
  int incy = 1;
  if (type->type_num == PyArray_DOUBLE)
    {
      double result;
      result = hipblasDdot(n, (double*)a_gpu,
			  incx, (double*)b_gpu, incy);
      gpaw_cublasSafeCall(cublasGetError());
      return PyFloat_FromDouble(result);
    }
  else
    {
      hipDoubleComplex result;
      result = hipblasZdotc(n, (hipDoubleComplex*)a_gpu,
			  incx, (hipDoubleComplex*)b_gpu, incy);
      gpaw_cublasSafeCall(cublasGetError());
      return PyComplex_FromDoubles(result.x,result.y);
    }

}


PyObject* dotu_cuda_gpu(PyObject *self, PyObject *args)
{
  hipDeviceptr_t a_gpu;
  hipDeviceptr_t b_gpu;
  
  PyObject *a_shape;
  PyArray_Descr *type;


  if (!PyArg_ParseTuple(args, "nOnO", &a_gpu,&a_shape,&b_gpu,&type))
    return NULL;
  int n = PyInt_AsLong(PyTuple_GetItem(a_shape,0));
  
  for (int i = 1; i < PyTuple_Size(a_shape); i++)
    n *= PyInt_AsLong(PyTuple_GetItem(a_shape,i));

  int incx = 1;
  int incy = 1;
  if (type->type_num == PyArray_DOUBLE)
    {
      double result;
      result = hipblasDdot(n, (double*)a_gpu,
			  incx, (double*)b_gpu, incy);
      gpaw_cublasSafeCall(cublasGetError());
      return PyFloat_FromDouble(result);
    }
  else
    {
      hipDoubleComplex result;
      result = hipblasZdotu(n, (hipDoubleComplex*)a_gpu,
			  incx, (hipDoubleComplex*)b_gpu, incy);
      gpaw_cublasSafeCall(cublasGetError());
      return PyComplex_FromDoubles(result.x,result.y);
    }
}

}
