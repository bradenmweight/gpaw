#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <float.h>

#include <sys/types.h>
#include <sys/time.h>

#include <hip/hip_complex.h>

#include "gpaw-cuda-int.h"

#ifndef MYJ
#ifndef CUGPAWCOMPLEX

__constant__ long c_offsets[FD_MAXCOEFS];
__constant__ double c_coefs[FD_MAXCOEFS];
__constant__ int c_offsets12[FD_MAXCOEFS];
__constant__ double c_coefs12[FD_MAXCOEFS];
__constant__ double c_coefs0[FD_MAXJ+1];

#endif
#endif

#undef  ACACHE_X
#undef BLOCK_X
#ifndef CUGPAWCOMPLEX
#define ACACHE_X  FD_ACACHE_X
#define BLOCK_X  FD_BLOCK_X
#else
#define ACACHE_X  FD_ACACHE_Xz
#define BLOCK_X  FD_BLOCK_Xz
#endif


#ifdef MYJ
#undef  FD_ACACHE_Y
#define FD_ACACHE_Y  ((FD_BLOCK_Y)+(MYJ))

#undef MYJ_X
#ifndef CUGPAWCOMPLEX
#define MYJ_X MYJ
#else
#define MYJ_X (2*(MYJ))
#endif

#undef CACHE_LOOP
#define CACHE_LOOP(i0e,i1s,i1e,i2s,i2e)					\
  for (c=0;c<MYJ;c++)							\
    acache0[c]=acache0[c+1];						\
  acache0[MYJ]=(i0e) ? MAKED(0) : a[(MYJ/2)*sizeyz];			\
  acache12p[0]=acache0[MYJ/2];						\
  if  ((i2tid<MYJ/2)) {							\
    acache12p[-MYJ/2]=(i2s) ? MAKED(0) : a[-MYJ/2];			\
    acache12p[FD_BLOCK_X]=(i2e) ? MAKED(0) : a[FD_BLOCK_X];		\
  }									\
  if  (i1tid<MYJ/2){							\
    acache12p[-FD_ACACHE_X*MYJ/2]=(i1s) ? MAKED(0) : a[-sizez*MYJ/2];	\
    acache12p[FD_ACACHE_X*FD_BLOCK_Y]=(i1e) ? MAKED(0) : a[sizez*FD_BLOCK_Y]; \
  }									\
  __syncthreads();							\
  x = MAKED(0);								\
  

#undef APPLY_LOOP
#define APPLY_LOOP(i0e)							\
  for (c = 0; c < ncoefs12; c++)					\
    IADD(x , MULTD(acache12p[c_offsets12[c]] , c_coefs12[c]));		\
  for (c = 0; c < MYJ+1; c++)						\
    IADD(x , MULTD(acache0[c] , c_coefs0[c]));				\
  if (!(i0e))								\
    b[0] = x;								\
  b+=c_n.y*c_n.z;							\
  a+=sizeyz;								\
  __syncthreads();							\

/*
__global__ void FD_kernel_bc(int ncoefs,int ncoefs12,int ncoefs0,
			     const Tcuda* a,Tcuda* b,const long3 c_n,
			     const long3 c_j,const int3 c_jb,int blocks)
{
  
  int i2bl=blockIdx.x/FD_XDIV;
  int xind=blockIdx.x-FD_XDIV*i2bl;
  int i2tid=threadIdx.x;
  int i2=i2bl*FD_BLOCK_X+i2tid;

  int i1bl=blockIdx.y/blocks;
  int blocksi=blockIdx.y-blocks*i1bl;

  int i1tid=threadIdx.y;
  int i1=i1bl*FD_BLOCK_Y+i1tid;

  __shared__ Tcuda acache12[FD_ACACHE_Y*FD_ACACHE_X];

  Tcuda acache0[MYJ+1];
  Tcuda x;
  int c;
  Tcuda *acache12p;
  int sizez=c_jb.z+c_n.z;  
  int sizeyz=c_j.y+c_n.y*sizez;
  
  int xlen=(c_n.x+FD_XDIV-1)/FD_XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_n.x);
  int xendbc=MIN(xend,c_n.x-MYJ/2);
  
  a+=(c_j.x+c_n.x*sizeyz)*blocksi;
  b+=(c_n.x*c_n.y*c_n.z)*blocksi;
  
  a+=xstart*sizeyz+i1*sizez+i2;
  b+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;

  acache12p=acache12+FD_ACACHE_X*(i1tid+MYJ/2)+i2tid+MYJ/2;

  for (c=1;c<MYJ+1;c++)
    acache0[c]=MAKED(0);
  
  int borders=0;
  if (i1bl==0)                     borders|=(1 << 0);
  if (i1bl==(gridDim.y-1)/blocks)  borders|=(1 << 1);
  if (i2bl==0)                     borders|=(1 << 2);
  if (i2bl==(gridDim.x-1)/FD_XDIV) borders|=(1 << 3); 

  if ((i2<c_n.z)&&(i1<c_n.y)) {
    for (c=1;c<MYJ+1;c++){
      int xind2=(c-1-MYJ/2);
      if ((xind2+xstart>=0)&&(xind2+xstart<c_n.x))
	acache0[c]=a[xind2*(sizeyz)];
    }
  }
  
  switch(borders){    
  case 0x0:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(0,0,0,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,0,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }
    break;

  case 0x1:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(0,1,0,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,1,0,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }    
    break;
    
  case 0x2:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(i1>=c_n.y,0,1,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i1>=c_n.y);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,1,0,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i1>=c_n.y);
    }    
    break;
  case 0x4:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(0,0,0,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,0,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }    
    break;
    
  case 0x5:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(0,1,0,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,1,0,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(0);
    }    
    break;
  case 0x6:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(i1>=c_n.y,0,1,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i1>=c_n.y);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,1,1,0);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i1>=c_n.y);
    }    
    break;
  case 0x8:    
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(i2>=c_n.z,0,0,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i2>=c_n.z);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,0,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i2>=c_n.z);
    }    
    break;
  case 0x9:    
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP(i2>=c_n.z,1,0,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i2>=c_n.z);
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,1,0,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP(i2>=c_n.z);
    }    
    break;
  case 0xA:    
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP((i2>=c_n.z)||(i1>=c_n.y),0,1,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP((i2>=c_n.z)||(i1>=c_n.y));
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,0,1,0,1);
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP((i2>=c_n.z)||(i1>=c_n.y));
    }    
    break;
  default:
    for (int i0=xstart; i0 < xendbc; i0++) {
      CACHE_LOOP((i2>=c_n.z)||(i1>=c_n.y),
	      ((i1-MYJ/2)<0),((i1+FD_BLOCK_Y)>=c_n.y),
	      ((i2-MYJ/2)<0),((i2+FD_BLOCK_X)>=c_n.z));
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP((i2>=c_n.z)||(i1>=c_n.y));
    }
    for (int i0=xendbc; i0 < xend ; i0++) {
      CACHE_LOOP(1,
	      ((i1-MYJ/2)<0),((i1+FD_BLOCK_Y)>=c_n.y),
	      ((i2-MYJ/2)<0),((i2+FD_BLOCK_X)>=c_n.z));
#include"fd-cuda-pragmas.cu"
      APPLY_LOOP((i2>=c_n.z)||(i1>=c_n.y));
    }    
    break;
  }
}
*/

__global__ void FD_kernel(int ncoefs,int ncoefs12,const double* a,
			  double* b,const long3 c_n,const long3 c_j,
			  const int3 c_jb,int blocks)
{
  
  int i2bl=blockIdx.x/FD_XDIV;
  int xind=blockIdx.x-FD_XDIV*i2bl;
  int i2tid=threadIdx.x;
  int i2=i2bl*BLOCK_X+i2tid;

  int i1bl=blockIdx.y/blocks;
  int blocksi=blockIdx.y-blocks*i1bl;

  int i1tid=threadIdx.y;
  int i1=i1bl*FD_BLOCK_Y+i1tid;

  __shared__ double acache12[FD_ACACHE_Y*ACACHE_X];

  double acache0[MYJ+1];
  double x;
  int c;
  double *acache12p;
  int sizez=c_jb.z+c_n.z;  
  int sizeyz=c_j.y+c_n.y*sizez;
  
  int xlen=(c_n.x+FD_XDIV-1)/FD_XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_n.x);
  
  a+=(c_j.x+c_n.x*sizeyz)*blocksi;
  b+=(c_n.x*c_n.y*c_n.z)*blocksi;
  
  a+=xstart*sizeyz+i1*sizez+i2;
  b+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;
  acache12p=acache12+ACACHE_X*(i1tid+MYJ/2)+i2tid+MYJ_X/2;
  for (c=1;c<MYJ+1;c++){
    acache0[c]=a[(c-1-MYJ/2)*(sizeyz)];
  }
  for (int i0=xstart; i0 < xend; i0++) {  
    for (c=0;c<MYJ;c++){
      acache0[c]=acache0[c+1];
    }   
    acache0[MYJ]=a[(MYJ/2)*sizeyz];

    acache12p[0]=acache0[MYJ/2];
    if  (i2tid<MYJ_X/2){
      acache12p[-MYJ_X/2]=a[-MYJ_X/2];
      acache12p[BLOCK_X]=a[BLOCK_X];
    }
    if  (i1tid<MYJ/2){
      acache12p[-ACACHE_X*MYJ/2]=a[-sizez*MYJ/2];
      acache12p[ACACHE_X*FD_BLOCK_Y]=a[sizez*FD_BLOCK_Y];      
    }
    __syncthreads();         
    x = 0.0;
#include"fd-cuda-pragmas.cu"
    for (c = 0; c < ncoefs12; c++){
      x+=acache12p[c_offsets12[c]]*c_coefs12[c];
    }	
    for (c = 0; c < MYJ+1; c++){
      x+=acache0[c]*c_coefs0[c];
    }	    
    for (c = 0; c < ncoefs; c++){	  
      x+=a[c_offsets[c]]*c_coefs[c];
    }	

    if ((i1<c_n.y) && (i2<c_n.z)) {
      b[0] = x;
    }

    b+=c_n.y*c_n.z;
    a+=sizeyz;
    __syncthreads();         
  }
  
}



/*
__global__ void FD_kernel(int ncoefs,int ncoefs12,int ncoefs0,const Tcuda* a,
			  Tcuda* b,const long3 c_n,const long3 c_j,
			  const int3 c_jb,int blocks)
{
  
  int i2bl=blockIdx.x/FD_XDIV;
  int xind=blockIdx.x-FD_XDIV*i2bl;
  int i2tid=threadIdx.x;
  int i2=i2bl*FD_BLOCK_X+i2tid;

  int i1bl=blockIdx.y/blocks;
  int blocksi=blockIdx.y-blocks*i1bl;

  int i1tid=threadIdx.y;
  int i1=i1bl*FD_BLOCK_Y+i1tid;

  __shared__ Tcuda acache12[FD_ACACHE_Y*FD_ACACHE_X];

  Tcuda acache0[MYJ+1];
  Tcuda x;
  int c;
  Tcuda *acache12p;
  int sizez=c_jb.z+c_n.z;  
  int sizeyz=c_j.y+c_n.y*sizez;
  
  int xlen=(c_n.x+FD_XDIV-1)/FD_XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_n.x);
  
  a+=(c_j.x+c_n.x*sizeyz)*blocksi;
  b+=(c_n.x*c_n.y*c_n.z)*blocksi;
  
  a+=xstart*sizeyz+i1*sizez+i2;
  b+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;

  acache12p=acache12+FD_ACACHE_X*(i1tid+MYJ/2)+i2tid+MYJ/2;
  for (c=1;c<MYJ+1;c++){
    acache0[c]=a[(c-1-MYJ/2)*(sizeyz)];
  }
  for (int i0=xstart; i0 < xend; i0++) {  
    for (c=0;c<MYJ;c++){
      acache0[c]=acache0[c+1];
    }   
    acache0[MYJ]=a[(MYJ/2)*sizeyz];

    acache12p[0]=acache0[MYJ/2];
    if  (i2tid<MYJ/2){
      acache12p[-MYJ/2]=a[-MYJ/2];
      acache12p[FD_BLOCK_X]=a[FD_BLOCK_X];
    }
    if  (i1tid<MYJ/2){
      acache12p[-FD_ACACHE_X*MYJ/2]=a[-sizez*MYJ/2];
      acache12p[FD_ACACHE_X*FD_BLOCK_Y]=a[sizez*FD_BLOCK_Y];      
    }
    __syncthreads();         
    x = MAKED(0.0);
#include"fd-cuda-pragmas.cu"
    for (c = 0; c < ncoefs12; c++){
      IADD(x , MULTD(acache12p[c_offsets12[c]] , c_coefs12[c]));
    }	
    for (c = 0; c < MYJ+1; c++){	  
      IADD(x , MULTD(acache0[c] , c_coefs0[c]));
    }	    
    for (c = 0; c < ncoefs; c++){	  
      IADD(x , MULTD(a[c_offsets[c]] , c_coefs[c]));
    }	

    if ((i1<c_n.y) && (i2<c_n.z)) {
      b[0] = x;
    }

    b+=c_n.y*c_n.z;
    a+=sizeyz;
    __syncthreads();         
  }
  
}
*/


#else
#define MYJ  2
#  define FD_kernel Zcuda(fd_kernel2)
#  define FD_kernel_bc Zcuda(fd_kernel2_bc)
#  include "fd-cuda.cu"
#  undef FD_kernel
#  undef FD_kernel_bc
#  undef MYJ
#define MYJ  4
#  define FD_kernel Zcuda(fd_kernel4)
#  define FD_kernel_bc Zcuda(fd_kernel4_bc)
#  include "fd-cuda.cu"
#  undef FD_kernel
#  undef FD_kernel_bc
#  undef MYJ
#define MYJ  6
#  define FD_kernel Zcuda(fd_kernel6)
#  define FD_kernel_bc Zcuda(fd_kernel6_bc)
#  include "fd-cuda.cu"
#  undef FD_kernel
#  undef FD_kernel_bc
#  undef MYJ
#define MYJ  8
#  define FD_kernel Zcuda(fd_kernel8)
#  define FD_kernel_bc Zcuda(fd_kernel8_bc)
#  include "fd-cuda.cu"
#  undef FD_kernel
#  undef FD_kernel_bc
#  undef MYJ
#define MYJ  10
#  define FD_kernel Zcuda(fd_kernel10)
#  define FD_kernel_bc Zcuda(fd_kernel10_bc)
#  include "fd-cuda.cu"
#  undef FD_kernel
#  undef FD_kernel_bc
#  undef MYJ


extern "C" {

  bmgsstencil_gpu bmgs_stencil_to_gpu(const bmgsstencil* s);

  void Zcuda(bmgs_fd_cuda_gpu)(const bmgsstencil_gpu* s_gpu, 
			       const Tcuda* adev, Tcuda* bdev,int blocks)  
  {
    int3 jb;
    

    long3 hc_n;
    long3 hc_j; 
    long* offsets_gpu;
    int* offsets12_gpu;

    hc_n.x=s_gpu->n[0];    hc_n.y=s_gpu->n[1];    hc_n.z=s_gpu->n[2];
    hc_j.x=s_gpu->j[0];    hc_j.y=s_gpu->j[1];    hc_j.z=s_gpu->j[2];

    adev+=(hc_j.x+hc_j.y+hc_j.z)/2;
	
#ifdef CUGPAWCOMPLEX
    hc_n.z*=2;
    hc_j.x*=2;    hc_j.y*=2;    hc_j.z*=2;
    offsets_gpu=s_gpu->offsets_gpuz;
    offsets12_gpu=s_gpu->offsets12_gpuz;
#else
    offsets_gpu=s_gpu->offsets_gpu;
    offsets12_gpu=s_gpu->offsets12_gpu;
#endif

    jb.z=hc_j.z;
    jb.y=hc_j.y/(hc_j.z+hc_n.z);

    if (s_gpu->ncoefs>0){
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets),offsets_gpu,
					   sizeof(long)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs),s_gpu->coefs_gpu,
					   sizeof(double)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
    }
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets12),offsets12_gpu,
					 sizeof(int)*s_gpu->ncoefs12,0,
					 hipMemcpyDeviceToDevice));
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs12),s_gpu->coefs12_gpu,
					 sizeof(double)*s_gpu->ncoefs12,0,
					 hipMemcpyDeviceToDevice));
    /*gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets0),s_gpu->offsets0_gpu,
					 sizeof(int)*s_gpu->ncoefs0,0,
					 hipMemcpyDeviceToDevice));*/
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs0),s_gpu->coefs0_gpu,
					 sizeof(double)*s_gpu->ncoefs0,0,
					 hipMemcpyDeviceToDevice));


    int gridx=FD_XDIV*MAX((hc_n.z+BLOCK_X-1)/BLOCK_X,1);
    int gridy=blocks*MAX((hc_n.y+FD_BLOCK_Y-1)/FD_BLOCK_Y,1);
    
    dim3 dimBlock(BLOCK_X,FD_BLOCK_Y); 
    dim3 dimGrid(gridx,gridy);    

    if (s_gpu->ncoefs0<=3)
      Zcuda(fd_kernel2)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12, (double*)adev,(double*)bdev, 
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=5)
      Zcuda(fd_kernel4)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12, (double*)adev,(double*)bdev, 
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=7)
      Zcuda(fd_kernel6)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12, (double*)adev,(double*)bdev, 
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=9)
      Zcuda(fd_kernel8)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,(double*)adev,(double*)bdev, 
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=11)
      Zcuda(fd_kernel10)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12, (double*)adev,(double*)bdev, 
	 hc_n,hc_j,jb,blocks);    
    gpaw_cudaSafeCall(hipGetLastError());

  }
  



  double Zcuda(bmgs_fd_cuda_cpu)(const bmgsstencil* s, const Tcuda* a, Tcuda* b)
  {
  
    Tcuda *adev,*bdev;

    size_t asize,bsize;
    struct timeval  t0, t1; 
    double flops;
    bmgsstencil_gpu s_gpu=bmgs_stencil_to_gpu(s);


    asize=s->j[0]+s->n[0]*(s->j[1]+s->n[1]*(s->n[2]+s->j[2]));
    bsize=s->n[0]*s->n[1]*s->n[2];

    gpaw_cudaSafeCall(hipGetLastError());
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(Tcuda)*asize));

    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(Tcuda)*bsize));

   
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(Tcuda)*asize,
				 hipMemcpyHostToDevice));
    gpaw_cudaSafeCall(hipGetLastError());
    gettimeofday(&t0,NULL);  
    Zcuda(bmgs_fd_cuda_gpu)(&s_gpu, adev,bdev,1);
      
    
    hipDeviceSynchronize(); 
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(Tcuda)*bsize,
				 hipMemcpyDeviceToHost));
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));

    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 

    return flops;
  
  
  }
  /*
  void Zcuda(bmgs_fd_cuda_gpu_bc)(const bmgsstencil_gpu* s_gpu, 
				  const Tcuda* adev, Tcuda* bdev,int blocks)  
  {
    int3 jb;
    

    long3 hc_n;
    long3 hc_j;    
    hc_n.x=s_gpu->n[0];    hc_n.y=s_gpu->n[1];    hc_n.z=s_gpu->n[2];
    //     hc_j.x=s_gpu->j[0];    hc_j.y=s_gpu->j[1];    hc_j.z=s_gpu->j[2];

    hc_j.x=0;    hc_j.y=0;    hc_j.z=0;
    
    jb.z=hc_j.z;
    jb.y=hc_j.y/(hc_j.z+hc_n.z);


    if (s_gpu->ncoefs>0){
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets),s_gpu->offsets_gpu,
					   sizeof(long)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs),s_gpu->coefs_gpu,
					   sizeof(double)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
    }
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets12),s_gpu->offsets12_gpu,
					 sizeof(int)*s_gpu->ncoefs12,0,
					 hipMemcpyDeviceToDevice));
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs12),s_gpu->coefs12_gpu,
					 sizeof(double)*s_gpu->ncoefs12,0,
					 hipMemcpyDeviceToDevice));
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs0),s_gpu->coefs0_gpu,
					 sizeof(double)*s_gpu->ncoefs0,0,
					 hipMemcpyDeviceToDevice));


    int gridx=FD_XDIV*MAX((s_gpu->n[2]+FD_BLOCK_X-1)/FD_BLOCK_X,1);
    int gridy=blocks*MAX((s_gpu->n[1]+FD_BLOCK_Y-1)/FD_BLOCK_Y,1);
    
    dim3 dimBlock(FD_BLOCK_X,FD_BLOCK_Y); 
    dim3 dimGrid(gridx,gridy);    

    adev+=(hc_j.x+hc_j.y+hc_j.z)/2;

    if (s_gpu->ncoefs0<=3)
      Zcuda(fd_kernel2_bc)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,s_gpu->ncoefs0,adev,bdev,
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=5)
      Zcuda(fd_kernel4_bc)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,s_gpu->ncoefs0,adev,bdev,
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=7)
      Zcuda(fd_kernel6_bc)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,s_gpu->ncoefs0,adev,bdev,
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=9)
      Zcuda(fd_kernel8_bc)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,s_gpu->ncoefs0,adev,bdev,
	 hc_n,hc_j,jb,blocks);    
    else if (s_gpu->ncoefs0<=11)
      Zcuda(fd_kernel10_bc)<<<dimGrid, dimBlock, 0>>>
	(s_gpu->ncoefs,s_gpu->ncoefs12,s_gpu->ncoefs0,adev,bdev,
	 hc_n,hc_j,jb,blocks);    
    gpaw_cudaSafeCall(hipGetLastError());

  }
*/
}

#ifndef CUGPAWCOMPLEX
#define CUGPAWCOMPLEX
#include "fd-cuda.cu"

extern "C" {

  bmgsstencil_gpu bmgs_stencil_to_gpu(const bmgsstencil* s)  
  {
    bmgsstencil_gpu s_gpu;/*
={s->ncoefs,NULL,NULL,0,NULL,NULL,0,NULL,NULL,
			   {s->n[0],s->n[1],s->n[2]},
			   {s->j[0],s->j[1],s->j[2]}};
			  */
    long offsets[s->ncoefs];
    int  offsets12[s->ncoefs];
    long offsetsz[s->ncoefs];
    int  offsets12z[s->ncoefs];
    int  offsets0[FD_MAXJ+1];
    double coefs[s->ncoefs],coefs12[s->ncoefs],coefs0[FD_MAXJ+1];
    long ncoefs=0,ncoefs12=0,ncoefs0=0;

    int n2=(s->n[2]+s->j[2]);
    int n1=s->j[1]+s->n[1]*n2;

    int jb[3];
    
    jb[2]=s->j[2];
    jb[1]=s->j[1]/n2;
    jb[0]=s->j[0]/n1;

        
    s_gpu.n[0]=s->n[0];    s_gpu.n[1]=s->n[1];    s_gpu.n[2]=s->n[2];
    s_gpu.j[0]=s->j[0];    s_gpu.j[1]=s->j[1];    s_gpu.j[2]=s->j[2];
    
    memset(coefs0,0,sizeof(double)*(FD_MAXJ+1));
    memset(offsets0,0,sizeof(int)*(FD_MAXJ+1));
    
    /*    fprintf(stdout,"%ld\t", s->ncoefs);
    for(int i = 0; i < s->ncoefs; ++i)
      fprintf(stdout,"(%d %lf %d)\t",i, s->coefs[i], s->offsets[i]);
      fprintf(stdout,"\n");
    fprintf(stdout,"\n%d %d %d %d %d %d\n",jb[0],jb[1],jb[2],s->n[0],s->n[1],s->n[2]);
    */
    
    
    for(int i = 0; i < s->ncoefs; i++){
      int offpoint=s->offsets[i]+(s->j[0]+s->j[1]+s->j[2])/2;
      int i0=offpoint/n1;
      int i1=(offpoint-i0*n1)/n2;
      int i2=(offpoint-i0*n1-i1*n2);
      i0-=jb[0]/2;
      i1-=jb[1]/2;
      i2-=jb[2]/2;
      //printf("%d %d i0 %d %d i1 %d  %d i2 %d %d\n",i,s->offsets[i],i0,jb[0],i1,jb[1],i2,jb[2]);
      if (i1==0 && i2==0 && abs(i0)<=jb[0]/2){
	//printf("of0 %d i0 %d %d i1 %d  %d i2 %d %d\n",i,i0,jb[0],i1,jb[1],i2,jb[2]);
	int offset=FD_MAXJ/2+i0;
	if (fabs(s->coefs[i]) > DBL_EPSILON){
	  offsets0[offset]=offset;
	  coefs0[offset]=s->coefs[i];
	  ncoefs0=MAX(ncoefs0,2*abs(i0)+1);	
	}
      } else if (i0==0 && abs(i1)<=jb[1]/2 && abs(i2)<=jb[2]/2 && (i1==0 || i2==0)){
	//printf("n12 %d i0 %d %d i1 %d  %d i2 %d %d\n",i,i0,jb[0],i1,jb[1],i2,jb[2]);
	offsets12[ncoefs12]=i2+FD_ACACHE_X*i1;
	offsets12z[ncoefs12]=2*i2+FD_ACACHE_Xz*i1;
      
	coefs12[ncoefs12]=s->coefs[i];
	ncoefs12++;	
      } else{
	offsets[ncoefs]=s->offsets[i];
	offsetsz[ncoefs]=2*s->offsets[i];
	coefs[ncoefs]=s->coefs[i];
	ncoefs++;
      }
    }
    ncoefs0=jb[0]+1;
    for(int i = 0; i < ncoefs0; i++){
      offsets0[i]=i;
      coefs0[i]=coefs0[i+(FD_MAXJ-ncoefs0+1)/2];
    }
    /*
    fprintf(stdout,"ncoefs %d\t", ncoefs);
    for(int i = 0; i < ncoefs; ++i)
      fprintf(stdout,"(%lf %d)\t", coefs[i], offsets[i]);
    fprintf(stdout,"\n");
    fprintf(stdout,"ncoefs0 %d\t", ncoefs0);
    for(int i = 0; i < ncoefs0; ++i)
      fprintf(stdout,"(%lf %d)\t", coefs0[i], offsets0[i]);
    fprintf(stdout,"\n");
    fprintf(stdout,"ncoefs12 %d\t", ncoefs12);
    for(int i = 0; i < ncoefs12; ++i)
      fprintf(stdout,"(%lf %d)\t", coefs12[i], offsets12[i]);
    fprintf(stdout,"\n");
    */
    s_gpu.ncoefs=ncoefs;
    s_gpu.ncoefs12=ncoefs12;
    s_gpu.ncoefs0=ncoefs0;

    s_gpu.coef_relax=s->coefs[0];

    if (ncoefs>0){
      GPAW_CUDAMALLOC(&(s_gpu.coefs_gpu),double,ncoefs);
      GPAW_CUDAMEMCPY(s_gpu.coefs_gpu,coefs,double,ncoefs, 
		      hipMemcpyHostToDevice);
      
      GPAW_CUDAMALLOC(&(s_gpu.offsets_gpu),long,ncoefs);
      GPAW_CUDAMEMCPY(s_gpu.offsets_gpu,offsets,long,ncoefs,
		      hipMemcpyHostToDevice);

      GPAW_CUDAMALLOC(&(s_gpu.offsets_gpuz),long,ncoefs);
      GPAW_CUDAMEMCPY(s_gpu.offsets_gpuz,offsetsz,long,ncoefs,
		      hipMemcpyHostToDevice);
    }
    GPAW_CUDAMALLOC(&(s_gpu.coefs12_gpu),double,ncoefs12);
    GPAW_CUDAMEMCPY(s_gpu.coefs12_gpu,coefs12,double,ncoefs12,
		    hipMemcpyHostToDevice);

    GPAW_CUDAMALLOC(&(s_gpu.offsets12_gpu),int,ncoefs12);
    GPAW_CUDAMEMCPY(s_gpu.offsets12_gpu,offsets12,int,ncoefs12,
		    hipMemcpyHostToDevice);

    GPAW_CUDAMALLOC(&(s_gpu.offsets12_gpuz),int,ncoefs12);
    GPAW_CUDAMEMCPY(s_gpu.offsets12_gpuz,offsets12z,int,ncoefs12,
		    hipMemcpyHostToDevice);

    GPAW_CUDAMALLOC(&(s_gpu.coefs0_gpu),double,ncoefs0);
    GPAW_CUDAMEMCPY(s_gpu.coefs0_gpu,coefs0,double,ncoefs0,
		    hipMemcpyHostToDevice);

    GPAW_CUDAMALLOC(&(s_gpu.offsets0_gpu),int,ncoefs0);
    GPAW_CUDAMEMCPY(s_gpu.offsets0_gpu,offsets0,int,ncoefs0,
		    hipMemcpyHostToDevice);

    return s_gpu;
  }

}


extern "C" {
  /* double bmgs_fd_cuda_cpu(const bmgsstencil* s, const double* a, double* b)
  {
  
    double *adev,*bdev;

    size_t asize,bsize;
    struct timeval  t0, t1; 
    double flops;
    bmgsstencil_gpu s_gpu=bmgs_stencil_to_gpu(s);


    asize=s->j[0]+s->n[0]*(s->j[1]+s->n[1]*(s->n[2]+s->j[2]));
    bsize=s->n[0]*s->n[1]*s->n[2];

    gpaw_cudaSafeCall(hipGetLastError());
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));

    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));

   
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    gpaw_cudaSafeCall(hipGetLastError());
    gettimeofday(&t0,NULL);  
    bmgs_fd_cuda_gpu(&s_gpu, adev,bdev,1);
      
    
    hipDeviceSynchronize(); 
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));

    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 

    return flops;
  
  
    }*/
  /*
  double bmgs_fd_cuda_cpu_bc(const bmgsstencil* s, const double* a, double* b)
  {
  
    double *adev,*bdev;

    size_t asize,bsize;
    struct timeval  t0, t1; 
    double flops;
    bmgsstencil_gpu s_gpu=bmgs_stencil_to_gpu(s);


    bsize=s->n[0]*s->n[1]*s->n[2];

    gpaw_cudaSafeCall(hipGetLastError());
    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*bsize));

    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));

   
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*bsize,
				 hipMemcpyHostToDevice));
    gpaw_cudaSafeCall(hipGetLastError());
    gettimeofday(&t0,NULL);  
    bmgs_fd_cuda_gpu_bc(&s_gpu, adev,bdev,1);
      
    
    hipDeviceSynchronize(); 
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);
    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));

    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 

    return flops;
  
  
  }
  */

}




#endif
#endif
