#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <sys/types.h>
#include <sys/time.h>

#include "gpaw-cuda-int.h"


#ifndef MYJ

/*__constant__ long c_offsets[FD_MAXCOEFS];
__constant__ double c_coefs[FD_MAXCOEFS];
__constant__ int c_offsets12[FD_MAXCOEFS];
__constant__ double c_coefs12[FD_MAXCOEFS];
__constant__ double c_coefs0[FD_MAXJ+1];
*/
#endif

#undef BLOCK_Y_B
#undef BLOCK_X_B
#ifdef MYJ	
#undef  FD_ACACHE_Y 
#define FD_ACACHE_Y  (FD_BLOCK_Y+MYJ)
#define BLOCK_X_B   FD_BLOCK_X_B
#define BLOCK_Y_B   FD_BLOCK_Y_B   

__global__ void RELAX_kernel(const int relax_method,const double coef_relax,
			     const int ncoefs,const double *c_coefs,
			     const long *c_offsets,
			     const int ncoefs12,const double *c_coefs12,
			     const int *c_offsets12,const double *c_coefs0,
			     const double* a,double* b,
			     const double* src,const long3  c_n,
			     const int3 c_jb,const int3 c_bjb,
			     const double w)
{
  int xx=gridDim.x/FD_XDIV;

  int xind=blockIdx.x/xx;
  int i2bl=blockIdx.x-xind*xx;
  
  int i2tid=threadIdx.x;
  int i2=i2bl*FD_BLOCK_X+i2tid;

  int i1tid=threadIdx.y;
  int i1=blockIdx.y*FD_BLOCK_Y+i1tid;

  __shared__ Tcuda acache12[FD_ACACHE_Y*FD_ACACHE_X];

  Tcuda acache0[MYJ+1];
  Tcuda *acache12p;
  int sizez=c_jb.z+c_n.z;  
  int sizeyz=(c_jb.y+c_n.y)*sizez;
  int sizebz=c_bjb.z+c_n.z;  
  int sizebyz=(c_bjb.y+c_n.y)*sizebz;

  int xlen=(c_n.x+FD_XDIV-1)/FD_XDIV;
  int xstart=xind*xlen;
  int xend=MIN(xstart+xlen,c_n.x);

  a+=xstart*sizeyz+i1*sizez+i2;
  b+=xstart*sizebyz+i1*sizebz+i2;
  src+=xstart*sizebyz+i1*sizebz+i2;
  //b+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;
  // src+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;
  

  acache12p=acache12+FD_ACACHE_X*(i1tid+MYJ/2)+i2tid+MYJ/2;
  
	
  if (relax_method == 1)
    {			
      /* Weighted Gauss-Seidel relaxation for the equation "operator" b = src
	 a contains the temporary array holding also the boundary values. */
      
      // Coefficient needed multiple times later
      //      const double coef = 1.0/c_coefs[0];
      
      /*NOT WORKIN ATM*/
      return;
    }
  else
    {
      /* Weighted Jacobi relaxation for the equation "operator" b = src
	 a contains the temporariry array holding also the boundary values. */

      for (int c=1;c<MYJ+1;c++){
	acache0[c]=a[(c-1-MYJ/2)*(sizeyz)];
      }
      for (int i0=xstart; i0 < xend; i0++) {  
	for (int c=0;c<MYJ;c++){
	  acache0[c]=acache0[c+1];
	}
	if ((i1<c_n.y+MYJ/2) && (i2<c_n.z+MYJ/2))
	  acache0[MYJ]=a[(MYJ/2)*sizeyz];
	
	acache12p[0]=acache0[MYJ/2];
	if  (i2tid<MYJ/2){
	  acache12p[-MYJ/2]=a[-MYJ/2];
	  acache12p[FD_BLOCK_X]=a[FD_BLOCK_X];
	}
	if  (i1tid<MYJ/2){
	  acache12p[-FD_ACACHE_X*MYJ/2]=a[-sizez*MYJ/2];
	  acache12p[FD_ACACHE_X*FD_BLOCK_Y]=a[sizez*FD_BLOCK_Y];      
	}
	__syncthreads();         
	Tcuda x = MAKED(0.0);	      
#if   MYJ==2
#pragma unroll 4
#elif MYJ==4
#pragma unroll 8
#elif MYJ==6
#pragma unroll 12
#elif MYJ==8
#pragma unroll 16
#elif MYJ==10
#pragma unroll 20
#endif
	for (int c = 0; c < ncoefs12; c++){
	  IADD(x , MULTD(acache12p[c_offsets12[c]] , c_coefs12[c]));
	}	
	for (int c = 0; c < MYJ/2; c++){	  
	  IADD(x , MULTD(acache0[c] , c_coefs0[c]));
	}	    
	for (int c = MYJ/2+1; c < MYJ+1; c++){	  
	  IADD(x , MULTD(acache0[c] , c_coefs0[c]));
	}	    
	for (int c = 0; c < ncoefs; c++){	  
	  IADD(x , MULTD(a[c_offsets[c]] , c_coefs[c]));
	}	
	
	if ((i1<c_n.y) && (i2<c_n.z)) {
	  b[0] = (1.0 - w) * b[0] + w * (src[0] - x)/coef_relax;
	  
	}
	b+=sizebyz;
	src+=sizebyz;
	//b+=c_n.y*c_n.z;
	//src+=c_n.y*c_n.z;
	a+=sizeyz;
	__syncthreads();         	
      }
      
    }

}



__global__ void RELAX_kernel_onlyb(const int relax_method,
				   const double coef_relax,
				   const int ncoefs,const double *c_coefs,
				   const long *c_offsets,
				   const int ncoefs12,const double *c_coefs12,
				   const int *c_offsets12,
				   const double *c_coefs0,
				   const double* a,double* b,
				   const double* src,const long3  c_n,
				   const int3 c_jb,const int boundary,
				   const double w)
{
  int xx=MAX((c_n.z+BLOCK_X_B-1)/BLOCK_X_B,1);
  int yy=MAX((c_n.y+BLOCK_Y_B-1)/BLOCK_Y_B,1);
  int ysiz=c_n.y;
  if ((boundary & GPAW_BOUNDARY_Y0) != 0) 
    ysiz-=BLOCK_Y_B;
  //ysiz-=c_jb.y/2;
  if ((boundary & GPAW_BOUNDARY_Y1) != 0) 
    ysiz-=BLOCK_Y_B;
  //ysiz-=c_jb.y/2;
  int yy2=MAX((ysiz+BLOCK_Y_B-1)/BLOCK_Y_B,0);

  int i2bl,i1bl;
  int xlen=c_n.x;
  int xind=0;
  int xstart=0;
  int i2pitch=0,i1pitch=0;
  int ymax=c_n.y,zmax=c_n.z,xmax=c_n.x;
  int xend,blockix;

  blockix=blockIdx.x;

  if ((boundary & GPAW_BOUNDARY_X0) != 0) {
    if ((blockix>=0) && (blockix<xx*yy)) {
      i1bl=blockix/xx;
      i2bl=blockix-i1bl*xx;
      xlen=c_jb.x/2;
      xstart=0;
    }
    blockix-=xx*yy;
  }
  if ((boundary & GPAW_BOUNDARY_X1) != 0) {
    if ((blockix>=0) && (blockix<xx*yy)) {
      i1bl=blockix/xx;
      i2bl=blockix-i1bl*xx;
      xlen=c_jb.x/2;
      xstart+=c_n.x-c_jb.x/2;
    }
    blockix-=xx*yy;    
  }
  if (blockix>=0){
    if ((boundary & GPAW_BOUNDARY_Y0) != 0) {
      if ((blockix>=0) && (blockix<FD_XDIV_B*xx)) {
	xind=blockix/xx;
	i2bl=blockix-xind*xx;
	i1bl=0;
	ymax=MIN(BLOCK_Y_B,ymax);
	//ymax=MIN(c_jb.y/2,ymax);
      }
      blockix-=FD_XDIV_B*xx;
    }
    if ((boundary & GPAW_BOUNDARY_Y1) != 0) {
      if ((blockix>=0) && (blockix<FD_XDIV_B*xx)) {
	xind=blockix/xx;
	i2bl=blockix-xind*xx;
	i1bl=0;
	//i1pitch=MAX(c_n.y-c_jb.y/2,0);
	i1pitch=MAX(c_n.y-BLOCK_Y_B,0);
      }
      blockix-=FD_XDIV_B*xx;
    }
    if ((boundary & GPAW_BOUNDARY_Z0) != 0) {
      if ((blockix>=0) && (blockix<FD_XDIV_B*yy2)) {
	xind=blockix/yy2;
	i2bl=0;
	zmax=MIN(BLOCK_X_B,zmax);
	//zmax=MIN(c_jb.z/2,zmax);
	i1bl=blockix-xind*yy2;
	if ((boundary & GPAW_BOUNDARY_Y0) != 0) 
	  i1pitch=BLOCK_Y_B;
	//i1pitch=c_jb.y/2;
	if ((boundary & GPAW_BOUNDARY_Y1) != 0) 
	  ymax=MAX(c_n.y-BLOCK_Y_B,0);
	//ymax=MAX(c_n.y-c_jb.y/2,0);	
      }
      blockix-=FD_XDIV_B*yy2;
    }
    if ((boundary & GPAW_BOUNDARY_Z1) != 0) {
      if ((blockix>=0) && (blockix<FD_XDIV_B*yy2)) {
	xind=blockix/yy2;
	i2bl=0;
	//i2pitch=MAX(c_n.z-c_jb.z/2,0);
	i2pitch=MAX(c_n.z-BLOCK_X_B,0);
	i1bl=blockix-xind*yy2;
	if ((boundary & GPAW_BOUNDARY_Y0) != 0) 
	  i1pitch=BLOCK_Y_B;
	//i1pitch=c_jb.y/2;
	if ((boundary & GPAW_BOUNDARY_Y1) != 0) 
	  ymax=MAX(c_n.y-BLOCK_Y_B,0);
	//ymax=MAX(c_n.y-c_jb.y/2,0);
      }
      blockix-=FD_XDIV_B*yy2;
    }
    if ((boundary & GPAW_BOUNDARY_X0) != 0) {
      xstart+=c_jb.x/2; 	
      xlen-=c_jb.x/2;
    }
    if ((boundary & GPAW_BOUNDARY_X1) != 0) {
      xlen-=c_jb.x/2;
      xmax-=c_jb.x/2;
    }
    xlen=(xlen+FD_XDIV_B-1)/FD_XDIV_B;
    xstart+=xind*xlen;        
  }
  xend=MIN(xstart+xlen,xmax);    
  if (blockix>=0){
    printf("Error!!\n");
    return;
  }
  

  int i2tid=threadIdx.x;
  int i2=i2pitch+i2bl*BLOCK_X_B+i2tid;

  int i1tid=threadIdx.y;
  int i1=i1pitch+i1bl*BLOCK_Y_B+i1tid;

  __shared__ double acache12[FD_ACACHE_Y*FD_ACACHE_X];

  double acache0[MYJ+1];
  double *acache12p;
  int sizez=c_jb.z+c_n.z;  
  int sizeyz=(c_jb.y+c_n.y)*sizez;


  acache12p=acache12+FD_ACACHE_X*(i1tid+MYJ/2)+i2tid+MYJ/2;

  a+=xstart*sizeyz+i1*sizez+i2;
  b+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;
  src+=xstart*c_n.y*c_n.z+i1*c_n.z+i2;
	
  if (relax_method == 1)
    {			
      /* Weighted Gauss-Seidel relaxation for the equation "operator" b = src
	 a contains the temporary array holding also the boundary values. */
      
      // Coefficient needed multiple times later
      //      const double coef = 1.0/c_coefs[0];
      
      /*NOT WORKIN ATM*/
      return;
    }
  else
    {
      /* Weighted Jacobi relaxation for the equation "operator" b = src
	 a contains the temporariry array holding also the boundary values. */

      for (int c=1;c<MYJ+1;c++){
	acache0[c]=a[(c-1-MYJ/2)*(sizeyz)];
      }
      for (int i0=xstart; i0 < xend; i0++) {  
	for (int c=0;c<MYJ;c++){
	  acache0[c]=acache0[c+1];
	}
	if ((i1<c_n.y+MYJ/2) && (i2<c_n.z+MYJ/2))
	  acache0[MYJ]=a[(MYJ/2)*sizeyz];

	acache12p[0]=acache0[MYJ/2];
	if  (i2tid<MYJ/2){
	  acache12p[-MYJ/2]=a[-MYJ/2];
	  acache12p[BLOCK_X_B]=a[BLOCK_X_B];
	}
	if  (i1tid<MYJ/2){
	  acache12p[-FD_ACACHE_X*MYJ/2]=a[-sizez*MYJ/2];
	  acache12p[FD_ACACHE_X*BLOCK_Y_B]=a[sizez*BLOCK_Y_B];      
	}
	__syncthreads();         
	Tcuda x = MAKED(0.0);	      
#if   MYJ==2
#pragma unroll 4
#elif MYJ==4
#pragma unroll 8
#elif MYJ==6
#pragma unroll 12
#elif MYJ==8
#pragma unroll 16
#elif MYJ==10
#pragma unroll 20
#endif
	for (int c = 0; c < ncoefs12; c++){
	  IADD(x , MULTD(acache12p[c_offsets12[c]] , c_coefs12[c]));
	}	
	for (int c = 0; c < MYJ/2; c++){	  
	  IADD(x , MULTD(acache0[c] , c_coefs0[c]));
	}	    
	for (int c = MYJ/2+1; c < MYJ+1; c++){	  
	  IADD(x , MULTD(acache0[c] , c_coefs0[c]));
	}	    
	for (int c = 0; c < ncoefs; c++){	  
	  IADD(x , MULTD(a[c_offsets[c]] , c_coefs[c]));
	}	
	
	if ((i1<ymax) && (i2<zmax)) {
	  b[0] = (1.0 - w) * b[0] + w * (src[0] - x)/coef_relax;
	  
	}
	b+=c_n.y*c_n.z;
	src+=c_n.y*c_n.z;
	a+=sizeyz;
	__syncthreads();         	
      }
      
    }

}


#else
#define MYJ  2
#  define RELAX_kernel relax_kernel2
#  define RELAX_kernel_onlyb relax_kernel2_onlyb
#  include "relax-cuda.cu"
#  undef RELAX_kernel
#  undef RELAX_kernel_onlyb
#  undef MYJ
#define MYJ  4
#  define RELAX_kernel relax_kernel4
#  define RELAX_kernel_onlyb relax_kernel4_onlyb
#  include "relax-cuda.cu"
#  undef RELAX_kernel
#  undef RELAX_kernel_onlyb
#  undef MYJ
#define MYJ  6
#  define RELAX_kernel relax_kernel6
#  define RELAX_kernel_onlyb relax_kernel6_onlyb
#  include "relax-cuda.cu"
#  undef RELAX_kernel
#  undef RELAX_kernel_onlyb
#  undef MYJ
#define MYJ  8
#  define RELAX_kernel relax_kernel8
#  define RELAX_kernel_onlyb relax_kernel8_onlyb
#  include "relax-cuda.cu"
#  undef RELAX_kernel
#  undef RELAX_kernel_onlyb
#  undef MYJ
#define MYJ  10
#  define RELAX_kernel relax_kernel10
#  define RELAX_kernel_onlyb relax_kernel10_onlyb
#  include "relax-cuda.cu"
#  undef RELAX_kernel
#  undef RELAX_kernel_onlyb
#  undef MYJ




extern "C" {


  bmgsstencil_gpu bmgs_stencil_to_gpu(const bmgsstencil* s);



  void bmgs_relax_cuda_gpu(const int relax_method,
			   const bmgsstencil_gpu* s_gpu, double* adev, 
			   double* bdev,const double* src, const double w,
			   int boundary,hipStream_t stream)
  {
    int3 jb;
    int3 bjb;
    int3 hc_bj;
    
    //jb.z=s_gpu->j[2];
    //jb.y=s_gpu->j[1]/(s_gpu->j[2]+s_gpu->n[2]);


    long3 hc_n;
    long3 hc_j;    
    hc_n.x=s_gpu->n[0];    hc_n.y=s_gpu->n[1];    hc_n.z=s_gpu->n[2];
    hc_j.x=s_gpu->j[0];    hc_j.y=s_gpu->j[1];    hc_j.z=s_gpu->j[2];

    bjb.x=0;    bjb.y=0;    bjb.z=0;
    hc_bj.x=0;    hc_bj.y=0;    hc_bj.y=0;
    
    /*
    if (s_gpu->ncoefs>0){
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets),s_gpu->offsets_gpu,
					   sizeof(long)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
      gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs),s_gpu->coefs_gpu,
					   sizeof(double)*s_gpu->ncoefs,0,
					   hipMemcpyDeviceToDevice));
    }
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets12),s_gpu->offsets12_gpu,
					  sizeof(int)*s_gpu->ncoefs12,0,
					  hipMemcpyDeviceToDevice));
    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs12),s_gpu->coefs12_gpu,
					 sizeof(double)*s_gpu->ncoefs12,0,
					 hipMemcpyDeviceToDevice));
    */
    /*    gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_offsets0),s_gpu->offsets0_gpu,
					 sizeof(int)*s_gpu->ncoefs0,0,
					 hipMemcpyDeviceToDevice));*/
    /*gpaw_cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(c_coefs0),s_gpu->coefs0_gpu,
					 sizeof(double)*s_gpu->ncoefs0,0,
					 hipMemcpyDeviceToDevice));
    */
    jb.z=hc_j.z;
    jb.y=hc_j.y/(hc_j.z+hc_n.z);
    jb.x=hc_j.x/((hc_j.z+hc_n.z)*hc_n.y+hc_j.y);
    
    if ((boundary & GPAW_BOUNDARY_SKIP) != 0) {
      int3 jb1;
      int3 bjb1,bjb2;
      bjb1.x=0;    bjb1.y=0;    bjb1.z=0;
      bjb2.x=0;    bjb2.y=0;    bjb2.z=0;      
      jb1.z=jb.z/2;
      jb1.x=jb.x/2;
      jb1.y=jb.y/2;
      if ((boundary & GPAW_BOUNDARY_X0) != 0) {
	bjb1.x+=jb.x/2;
      }
      if ((boundary & GPAW_BOUNDARY_X1) != 0) {
	bjb2.x+=jb.x/2;
      }
      if ((boundary & GPAW_BOUNDARY_Y0) != 0) {
	bjb1.y+=BLOCK_Y_B;
	//bjb1.y+=jb.y/2;
      }
      if ((boundary & GPAW_BOUNDARY_Y1) != 0) {
	bjb2.y+=BLOCK_Y_B;
	//bjb2.y+=jb.y/2;
      }
      if ((boundary & GPAW_BOUNDARY_Z0) != 0) {
	bjb1.z+=BLOCK_X_B;
	//bjb1.z+=jb.z/2;
      }
      if ((boundary & GPAW_BOUNDARY_Z1) != 0) {
	bjb2.z+=BLOCK_X_B;
	//bjb2.z+=jb.z/2;
      }
      bjb.x=bjb1.x+bjb2.x;
      bjb.y=bjb1.y+bjb2.y;
      bjb.z=bjb1.z+bjb2.z;

      hc_n.x-=bjb.x;    hc_n.y-=bjb.y;    hc_n.z-=bjb.z;

      jb.x+= bjb.x;    jb.y+=bjb.y;    jb.z+=bjb.z;
      jb1.x+= bjb1.x;    jb1.y+=bjb1.y;    jb1.z+=bjb1.z;
      
      hc_bj.z=bjb.z;
      hc_bj.y=bjb.y*(hc_bj.z+hc_n.z);
      hc_bj.x=bjb.x*((hc_bj.z+hc_n.z)*hc_n.y+hc_bj.y);
      
      hc_j.z=jb.z;
      hc_j.y=jb.y*(hc_j.z+hc_n.z);
      hc_j.x=jb.x*((hc_j.z+hc_n.z)*hc_n.y+hc_j.y);
      
      bdev+=bjb1.z+bjb1.y*(hc_bj.z+hc_n.z)+
	bjb1.x*((hc_bj.z+hc_n.z)*hc_n.y+hc_bj.y);
      src+=bjb1.z+bjb1.y*(hc_bj.z+hc_n.z)+
	bjb1.x*((hc_bj.z+hc_n.z)*hc_n.y+hc_bj.y);
      
      adev=(Tcuda*)((double*)adev+jb1.z+jb1.y*(hc_j.z+hc_n.z)+
		    jb1.x*((hc_j.z+hc_n.z)*hc_n.y+hc_j.y));

    }else {
      adev=(Tcuda*)((double*)adev +(hc_j.x+hc_j.y+hc_j.z)/2);
    }
    if ((hc_n.x<=0) || (hc_n.y<=0) || (hc_n.z<=0))
      return;
    
    dim3 dimBlock(1,1,1);
    dim3 dimGrid(1,1,1);
    if (((boundary & GPAW_BOUNDARY_NORMAL) != 0) ||
	((boundary & GPAW_BOUNDARY_SKIP) != 0)){
      dimGrid.x=FD_XDIV*MAX((hc_n.z+FD_BLOCK_X-1)/FD_BLOCK_X,1);
      dimGrid.y=MAX((hc_n.y+FD_BLOCK_Y-1)/FD_BLOCK_Y,1);
      dimBlock.x=FD_BLOCK_X;
      dimBlock.y=FD_BLOCK_Y;
    } else if ((boundary & GPAW_BOUNDARY_ONLY) != 0) {
      int xx=MAX((hc_n.z+BLOCK_X_B-1)/BLOCK_X_B,1);
      int yy=MAX((hc_n.y+BLOCK_Y_B-1)/BLOCK_Y_B,1);
      int ysiz=hc_n.y;
      if ((boundary & GPAW_BOUNDARY_Y0) != 0) 
	ysiz-=BLOCK_Y_B;
      //ysiz-=jb.y/2;
      if ((boundary & GPAW_BOUNDARY_Y1) != 0) 
	ysiz-=BLOCK_Y_B;
      //ysiz-=jb.y/2;
      int yy2=MAX((ysiz+BLOCK_Y_B-1)/BLOCK_Y_B,0);
      dimGrid.x=0;
      if ((boundary & GPAW_BOUNDARY_X0) != 0) 
	dimGrid.x+=xx*yy;
      if ((boundary & GPAW_BOUNDARY_X1) != 0) 
	dimGrid.x+=xx*yy;
      if ((boundary & GPAW_BOUNDARY_Y0) != 0) 
	dimGrid.x+=FD_XDIV_B*xx;
      if ((boundary & GPAW_BOUNDARY_Y1) != 0) 
	dimGrid.x+=FD_XDIV_B*xx;
      if ((boundary & GPAW_BOUNDARY_Z0) != 0) 
	dimGrid.x+=FD_XDIV_B*yy2;
      if ((boundary & GPAW_BOUNDARY_Z1) != 0) 
	dimGrid.x+=FD_XDIV_B*yy2;
      dimGrid.y=1;
      dimBlock.x=BLOCK_X_B;
      dimBlock.y=BLOCK_Y_B;

    }
    

    if (((boundary & GPAW_BOUNDARY_NORMAL) != 0) ||
	((boundary & GPAW_BOUNDARY_SKIP) != 0)){
      if (s_gpu->ncoefs0<=3)
	relax_kernel2<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,bjb,w);    
      else if (s_gpu->ncoefs0<=5)
	relax_kernel4<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,bjb,w);    
      else if (s_gpu->ncoefs0<=7)
	relax_kernel6<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,bjb,w);    
      else if (s_gpu->ncoefs0<=9)
	relax_kernel8<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,bjb,w);    
      else if (s_gpu->ncoefs0<=11)
	relax_kernel10<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,bjb,w);    
    } else if ((boundary & GPAW_BOUNDARY_ONLY) != 0) {
      if (s_gpu->ncoefs0<=3)
	relax_kernel2_onlyb<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,boundary,w);    
      else if (s_gpu->ncoefs0<=5)
	relax_kernel4_onlyb<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,boundary,w);    
      else if (s_gpu->ncoefs0<=7)
	relax_kernel6_onlyb<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,boundary,w);    
      else if (s_gpu->ncoefs0<=9)
	relax_kernel8_onlyb<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,boundary,w);    
      else if (s_gpu->ncoefs0<=11)
	relax_kernel10_onlyb<<<dimGrid, dimBlock, 0, stream>>>
	  (relax_method,s_gpu->coef_relax,
	   s_gpu->ncoefs,s_gpu->coefs_gpu,s_gpu->offsets_gpu,
	   s_gpu->ncoefs12,s_gpu->coefs12_gpu,s_gpu->offsets12_gpu,
	   s_gpu->coefs0_gpu,
	   adev,bdev,src,hc_n,jb,boundary,w); 
    }
    gpaw_cudaSafeCall(hipGetLastError());
  }


  double bmgs_relax_cuda_cpu(const int relax_method, const bmgsstencil* s,
			     double* a, double* b,const double* src, 
			     const double w)
  {
    double *adev,*bdev,*srcdev;
    size_t asize,bsize;
    struct timeval  t0, t1; 
    double flops;
    bmgsstencil_gpu s_gpu=bmgs_stencil_to_gpu(s);
    
    asize=s->j[0]+s->n[0]*(s->j[1]+s->n[1]*(s->n[2]+s->j[2]));
    bsize=s->n[0]*s->n[1]*s->n[2];

    gpaw_cudaSafeCall(hipMalloc(&adev,sizeof(double)*asize));
   
    gpaw_cudaSafeCall(hipMalloc(&bdev,sizeof(double)*bsize));
    gpaw_cudaSafeCall(hipMalloc(&srcdev,sizeof(double)*bsize));
   
    gpaw_cudaSafeCall(hipMemcpy(adev,a,sizeof(double)*asize,
				 hipMemcpyHostToDevice));
    gpaw_cudaSafeCall(hipMemcpy(bdev,b,sizeof(double)*bsize,
				 hipMemcpyHostToDevice));
    gpaw_cudaSafeCall(hipMemcpy(srcdev,src,sizeof(double)*bsize,
				 hipMemcpyHostToDevice));
   
    gettimeofday(&t0,NULL);
    bmgs_relax_cuda_gpu(relax_method, &s_gpu, adev, bdev,srcdev, w,
			GPAW_BOUNDARY_NORMAL,0);

    hipDeviceSynchronize();  
    gpaw_cudaSafeCall(hipGetLastError());

    gettimeofday(&t1,NULL);

    gpaw_cudaSafeCall(hipMemcpy(b,bdev,sizeof(double)*bsize,
				 hipMemcpyDeviceToHost));
    
    gpaw_cudaSafeCall(hipFree(adev));
    gpaw_cudaSafeCall(hipFree(bdev));
    gpaw_cudaSafeCall(hipFree(srcdev));

    flops=(t1.tv_sec*1.0+t1.tv_usec/1000000.0-t0.tv_sec*1.0-t0.tv_usec/1000000.0); 
   
    return flops;
  
  }
  
}
#endif
